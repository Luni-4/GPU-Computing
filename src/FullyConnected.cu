#include <iostream>
#include <vector>
#include <algorithm>

#ifdef _WIN32
#include "Windows.h"
#endif

// Funzioni comuni
#include "Common.h"

// Cuda Kernel
#include "Kernel.h"

// Classi
#include "FullyConnected.h"

FullyConnected::FullyConnected(const int &width, const int &height, const ActFctType &a)
	: LayerDefinition(width, height, 1, FULLY_CONNECTED, a) {

	this->_nodes = width * height;
	this->_alignedNodes = ALIGN_UP(_nodes);

}

FullyConnected::FullyConnected(const int &width, const ActFctType &a)
	: LayerDefinition(width, 1, 1, FULLY_CONNECTED, a),
	_nodes(width) {

	this->_alignedNodes = ALIGN_UP(_nodes);

}

FullyConnected::~FullyConnected() {

}


std::vector<double> FullyConnected::getWeights() {
	std::vector<double> wCPU(_wDim);
	CHECK(hipMemcpy(&wCPU[0], weight, _wDim * sizeof(double), hipMemcpyDeviceToHost));
	return wCPU;
}

std::vector<double> FullyConnected::getBias() {
	std::vector<double> bCPU(_nodes);
	CHECK(hipMemcpy(&bCPU[0], bias, _nodes * sizeof(double), hipMemcpyDeviceToHost));
	return bCPU;
}


void FullyConnected::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {

	// Dimensione matrice dei pesi
	_wDim = prevLayerWidth * prevLayerHeight * prevLayerDepth * _nodes;

	// Salvare dimensione del livello precedente
	_prevLayerDim = prevLayerWidth * prevLayerHeight * prevLayerDepth;

	// Dimensione matrice dei pesi in byte
	_wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);
	
	// Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));
	
#ifdef DEBUG
	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);
#endif

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, _wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	CHECK(hipMalloc((void**)&error, Bytes));
	CHECK(hipMalloc((void**)&temp, _wBytes));

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(prevLayerWidth * prevLayerHeight);

	// Tanti blocchi quanto sono i nodi e la profondità del layer precedente
	dim3 numBlocks(_nodes, prevLayerDepth, 1);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi del livello precedente
	dim3 threadBlocks(aligned, 1, 1);

	// Inizializza array per numeri casuali
	hiprandState *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _nodes * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandState)));

	// Inizializzare i weight del livello
	Kernel::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

	// Inizializzare i bias del livello
	Kernel::initBiasK(1, _alignedNodes, bias, _nodes, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore dei pesi\n\n";
	pettyPrintCuda(weight, _wDim, _prevLayerDim);
	std::cout << "\n\nValore dei bias\n\n";
	pettyPrintCuda(bias, _nodes, 1);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}


void FullyConnected::forward_propagation(const double *prevOutput) {

	CHECK_CUBLAS(
		hipblasDgemv(handle, HIPBLAS_OP_T, _prevLayerDim, _nodes, &alpha, weight, _prevLayerDim, prevOutput, 1, &beta, output, 1));

#ifdef DEBUG
	std::cout << "\n\nOutput dei nodi senza bias\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

	// Somma con il bias
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &alpha, bias, 1, output, 1));

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nOutput dei nodi con bias sommato\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif

	// Applicare funzione di attivazione
	if (_a == RELU)
		Kernel::actReluK(1, _alignedNodes, output, _nodes);
	else if (_a == SIGMOID)
		Kernel::actSigmoidK(1, _alignedNodes, output, _nodes);
	else if (_a == TANH)
		Kernel::actTanhK(1, _alignedNodes, output, _nodes);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nOutput dei nodi con funzione di attivazione\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif
}

void FullyConnected::back_propagation(const double *prevOutput, const double *forwardWeight, const double *forwardError, const int &forwardNodes, const double &learningRate) {
    
    // Propagazione dell'errore dal livello successivo
    CHECK_CUBLAS(
		hipblasDgemv(handle, HIPBLAS_OP_N, _nodes ,forwardNodes, &alpha, forwardWeight, _nodes, forwardError, 1, &beta, error, 1));
	
	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nForward weight\n\n";
	pettyPrintCuda(forwardWeight, _nodes * forwardNodes, forwardNodes);
	std::cout << "\n\nForward error\n\n";
	pettyPrintCuda(forwardError, forwardNodes, 1);
	std::cout << "\n\nErrore commesso sui nodi back propagation\n\n";
	pettyPrintCuda(error, _nodes, 1);
#endif
		
	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);   

}

void FullyConnected::back_propagation_output(const double *prevOutput, const uint8_t *labels, const int &target, const double &learningRate) {
    
    // Calcolo dell'errore per ogni nodo
    Kernel::outputErrorK(1, _alignedNodes, output, error, labels, target, _nodes);
    
    // CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
	
#ifdef DEBUG
	std::cout << "\n\nErrore commesso sui nodi back propagation output\n\n";
	pettyPrintCuda(error, _nodes, 1);
#endif
    
    // Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);  

}

void FullyConnected::deleteCuda() {

	CHECK_CUBLAS(hipblasDestroy(handle));
	CHECK(hipFree(weight));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
	CHECK(hipFree(temp));
}

void FullyConnected::calcBackPropagation(const double *prevOutput, const double &learningRate) {

    // Applicare derivata della funzione di attivazione
	if (_a == RELU)
		Kernel::derivActReluK(1, _alignedNodes, output, error, _nodes);
	else if (_a == SIGMOID)
		Kernel::derivActSigmoidK(1, _alignedNodes, output, error, _nodes);
	else if (_a == TANH)
		Kernel::derivActTanhK(1, _alignedNodes, output, error, _nodes);
		
	// CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());
    
#ifdef DEBUG
	std::cout << "\n\nErrore commesso sui nodi con relativa derivata\n\n";
	pettyPrintCuda(error, _nodes, 1);
#endif
    
    // Aggiornare i pesi (da mettere in funzione)    
    updateWeights(prevOutput, learningRate);
}

void FullyConnected::updateWeights(const double *prevOutput, const double &learningRate) {
	
	// Riempire la matrice temporanea di 0
	CHECK(hipMemset(temp, 0, _wBytes));

#ifdef DEBUG
	std::cout << "\n\nMatrice temporanea valore iniziale\n\n";
	pettyPrintCuda(temp, _wDim, _prevLayerDim);
#endif
	
	// Deve ricevere lo scalare dal device
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);		
	
	for (int i = 0; i < _nodes; i++){
	    CHECK_CUBLAS(
		    hipblasDaxpy(handle, _prevLayerDim, &error[i], prevOutput, 1, temp + (i * _prevLayerDim), 1));   
        
        // CPU deve attendere che esecuzione della funzione finisca
        CHECK(hipDeviceSynchronize());
    }

#ifdef DEBUG
	std::cout << "\n\nMatrice temporanea per aggiornamento pesi\n\n";
	pettyPrintCuda(temp, _wDim, _prevLayerDim);
#endif

    // Deve ricevere lo scalare dall'host
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);    
    
    // Aggiornamento effettivo dei pesi 
    CHECK_CUBLAS(
		hipblasDaxpy(handle, _wDim, &learningRate, temp, 1, weight, 1));
		
    // CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nMatrice dei pesi aggiornata\n\n";
	pettyPrintCuda(weight, _wDim, _prevLayerDim);
#endif
	
	// Aggiornamento del bias 
    CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &learningRate, error, 1, bias, 1));
		
    // CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nVettore del bias aggiornato\n\n";
	pettyPrintCuda(bias, _nodes, 1);
#endif
}
