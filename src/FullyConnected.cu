#include <iostream>

#ifdef _WIN32
#include "Windows.h"
#endif

// Funzioni comuni
#include "Common.h"

// Cuda Kernel
#include "Kernel.h"

// Classi
#include "FullyConnected.h"

FullyConnected::FullyConnected(const int &width, const int &height, const ActFctType &a)
	: LayerDefinition(width, height, 1, FULLY_CONNECTED, a) {

	this->_nodes = width * height;
	this->_alignedNodes = ALIGN_UP(_nodes, THREADS);


}

FullyConnected::FullyConnected(const int &width, const ActFctType &a)
	: LayerDefinition(width, 1, 1, FULLY_CONNECTED, a),
	_nodes(width) {

	this->_alignedNodes = ALIGN_UP(_nodes, THREADS);
}

FullyConnected::~FullyConnected() {

}


std::vector<double> FullyConnected::getWeights(void) {
	std::vector<double> wCPU(_wDim);
	CHECK(hipMemcpy(&wCPU[0], weight, _wBytes, hipMemcpyDeviceToHost));
	return wCPU;
}

std::vector<double> FullyConnected::getBias(void) {
	std::vector<double> bCPU(_nodes);
	CHECK(hipMemcpy(&bCPU[0], bias, _nodes * sizeof(double), hipMemcpyDeviceToHost));
	return bCPU;
}

int FullyConnected::getPredictionIndex(void) {
	int maxIndex;

	// Individuare indice (classe) che corrisponde al valore massimo di output
	CHECK_CUBLAS(
		hipblasIdamax(handle, _nodes, output, 1, &maxIndex));

	return maxIndex - 1;
}

void FullyConnected::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {

	// Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));

	// Impostazioni della cache
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	// Dimensione matrice dei pesi
	_wDim = prevLayerWidth * prevLayerHeight * prevLayerDepth * _nodes;

	// Salvare dimensione del livello precedente
	_prevLayerDim = prevLayerWidth * prevLayerHeight * prevLayerDepth;

	// Dimensione matrice dei pesi in byte
	_wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);

#ifdef DEBUG
	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);
#endif

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, _wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	CHECK(hipMalloc((void**)&error, Bytes));
	CHECK(hipMalloc((void**)&temp, _wBytes));

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(prevLayerWidth * prevLayerHeight, THREADS);

	// Tanti blocchi quanto sono i nodi e la profondità del layer precedente
	dim3 numBlocks(_nodes, prevLayerDepth, 1);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi del livello precedente
	dim3 threadBlocks(aligned, 1, 1);

	// Inizializza array per numeri casuali
	hiprandStateXORWOW_t *devStates;
	//hiprandState *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _nodes * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandStateXORWOW_t)));

	// Inizializzare i weight del livello
	Kernel::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// Inizializzare i bias del livello
	Kernel::initBiasK(_alignedNodes / THREADS, THREADS, bias, _nodes, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore dei pesi\n\n";
	pettyPrintCuda(weight, _wDim, _prevLayerDim);
	std::cout << "\n\nValore dei bias\n\n";
	pettyPrintCuda(bias, _nodes, 1);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}


void FullyConnected::forward_propagation(const double *prevOutput) {

	//std::cout << "\n\n\nFORWARD\n";
	//printW();

	CHECK_CUBLAS(
		hipblasDgemv(handle, HIPBLAS_OP_T, _prevLayerDim, _nodes, &alpha, weight, _prevLayerDim, prevOutput, 1, &beta, output, 1));

#ifdef DEBUG
	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nImmagine di input\n\n";
	pettyPrintCuda(prevOutput, _prevLayerDim, 1);
	std::cout << "\n\nOutput dei nodi senza bias\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif

	CHECK_CUBLAS(
		hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _nodes, &alpha, bias, 1, &alpha, output, 1, output, 1));

#ifdef DEBUG
	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nOutput dei nodi con bias sommato\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif    

	// Applicare funzione di attivazione
	if (_a == RELU)
		Kernel::actReluK(_alignedNodes / THREADS, THREADS, output, temp, _nodes);
	else if (_a == SIGMOID)
		Kernel::actSigmoidK(_alignedNodes / THREADS, THREADS, output, _nodes);
	else if (_a == TANH)
		Kernel::actTanhK(_alignedNodes / THREADS, THREADS, output, _nodes);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
    
#ifdef DEBUG
	std::cout << "\n\nOutput dei nodi con funzione di attivazione\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif
}

void FullyConnected::calcError(double *prevError, const int &prevNodes) {

	// Propagazione dell'errore dal livello successivo
	CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _nodes, prevNodes, &alpha, weight, _nodes, error, 1, &beta, prevError, 1));


#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nForward weight\n\n";
	pettyPrintCuda(weight, _wDim, prevNodes);
	std::cout << "\n\nForward error\n\n";
	pettyPrintCuda(error, _nodes, 1);
	std::cout << "\n\nErrore commesso sui nodi back propagation\n\n";
	pettyPrintCuda(prevError, prevNodes, 1);
#endif
}


void FullyConnected::back_propagation(const double *prevOutput, const double &learningRate) {

	// Aggiornare i pesi (da mettere in funzione)    
	calcBackPropagation(prevOutput, learningRate);

}

void FullyConnected::back_propagation_output(const double *prevOutput, const uint8_t *labels, const int &target, const double &learningRate) {

	// Calcolo dell'errore per ogni nodo
	Kernel::outputErrorK(_alignedNodes / THREADS, THREADS, output, error, labels, target, _nodes);

	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nErrore commesso sui nodi back propagation output\n\n";
	pettyPrintCuda(error, _nodes, 1);
#endif

	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);

}

inline void FullyConnected::calcBackPropagation(const double *prevOutput, const double &learningRate) {

	// Applicare derivata della funzione di attivazione
	if (_a == RELU)
		Kernel::derivActReluK(_alignedNodes / THREADS, THREADS, error, temp, _nodes);
	else if (_a == SIGMOID)
		Kernel::derivActSigmoidK(_alignedNodes / THREADS, THREADS, output, error, _nodes);
	else if (_a == TANH)
		Kernel::derivActTanhK(_alignedNodes / THREADS, THREADS, output, error, _nodes);


#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi con relativa derivata\n\n";
	pettyPrintCuda(error, _nodes, 1);
#endif

	// Aggiornare i pesi (da mettere in funzione)    
	updateWeights(prevOutput, learningRate);
}

void FullyConnected::updateWeights(const double *prevOutput, const double &learningRate) {

	int dim = ALIGN_UP(_nodes * _prevLayerDim, THREADS);

	Kernel::errorPrevOutputK(dim / THREADS, THREADS, temp, prevOutput, error, _nodes, _nodes * _prevLayerDim, _prevLayerDim);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice temporanea per aggiornamento pesi\n\n";
	pettyPrintCuda(temp, _wDim, _prevLayerDim);
#endif		

	CHECK_CUBLAS(hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, _nodes, _prevLayerDim, &learningRate, temp, _nodes, &alpha, weight, _nodes, weight, _nodes));

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice dei pesi aggiornata\n\n";
	pettyPrintCuda(weight, _wDim, _prevLayerDim);
#endif	
	
    CHECK_CUBLAS(hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _nodes, &learningRate, error, 1, &alpha, bias, 1, bias, 1));
		
    // CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());
    
#ifdef DEBUG
	std::cout << "\n\nVettore del bias aggiornato\n\n";
	pettyPrintCuda(bias, _nodes, 1);
#endif
}

void FullyConnected::deleteCuda(void) {

	CHECK_CUBLAS(hipblasDestroy(handle));
	CHECK(hipFree(weight));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
	CHECK(hipFree(temp));
}

void FullyConnected::printW() {
	printFromCudaFormatted(weight, _wDim, _prevLayerDim);
}