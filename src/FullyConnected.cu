#include <iostream>
#include <vector>
#include <algorithm>

#ifdef _WIN32
#include "Windows.h"
#endif

// Funzioni comuni
#include "Common.h"

// Cuda Kernel
#include "Kernel.h"

// Classi
#include "FullyConnected.h"

FullyConnected::FullyConnected(const int &width, const int &height, const ActFctType &a)
	: LayerDefinition(width, height, 1, FULLY_CONNECTED, a) {

	this->_nodes = width * height;		
	this->_alignedNodes = ALIGN_UP(_nodes);

}

FullyConnected::FullyConnected(const int &width, const ActFctType &a)
	: LayerDefinition(width, 1, 1, FULLY_CONNECTED, a),
	_nodes(width) {
	
	this->_alignedNodes = ALIGN_UP(_nodes);

}

FullyConnected::~FullyConnected() {

}

int FullyConnected::getLayerNodeCount() {
	return _nodes;
}


int FullyConnected::getWeightCount(const int &prevLayerNode) {
	return prevLayerNode * _nodes;
}


std::vector<double> FullyConnected::getWeights() {
	std::vector<double> wCPU(_wDim);
	CHECK(hipMemcpy(&wCPU[0], weight, _wDim * sizeof(double), hipMemcpyDeviceToHost));
	return wCPU;
}

std::vector<double> FullyConnected::getBias() {
	std::vector<double> bCPU(_nodes);
	CHECK(hipMemcpy(&bCPU[0], bias, _nodes * sizeof(double), hipMemcpyDeviceToHost));
	return bCPU;
}


void FullyConnected::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {

	// Dimensione matrice dei pesi
	_wDim = prevLayerWidth * prevLayerHeight * prevLayerDepth * _nodes;
	
	// Salvare dimensione del livello precedente
	_prevLayerDim = prevLayerWidth * prevLayerHeight * prevLayerDepth;  

	// Dimensione matrice dei pesi in byte
	const unsigned int wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);

	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	CHECK(hipMalloc((void**)&error, Bytes));

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(prevLayerWidth * prevLayerHeight);

	// Tanti blocchi quanto sono i nodi e la profondità del layer precedente
	dim3 numBlocks(_nodes, prevLayerDepth, 1);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi del livello precedente
	dim3 threadBlocks(aligned, 1, 1);

	// Inizializza array per numeri casuali
	hiprandState *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _nodes * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandState)));

	// Inizializzare i weight del livello
	Kernel::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

	// Inizializzare i bias del livello
	Kernel::initBiasK(1, _alignedNodes, bias, _nodes, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore dei pesi\n\n";
	printFromCudaFormatted(weight, _wDim, prevLayerWidth);
	std::cout << "\n\nValore dei bias\n\n";
	printFromCudaFormatted(bias, _nodes, 1);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}


void FullyConnected::forward_propagation(const double *prev) {

	// Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));

	// Fattori dei prodotti
	const double alpha = 1.0f;
	const double beta = 0.0f;

	CHECK_CUBLAS(
		hipblasDgemv(handle, HIPBLAS_OP_N, _prevLayerDim, _nodes, &alpha, weight, _prevLayerDim, prev, 1, &beta, output, 1));

    // CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());

	// Somma con il bias
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &alpha, bias, 1, output, 1));

    // CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nOutput dei nodi\n\n";
	printFromCuda(output, _nodes);
#endif

	// Applicare funzione di attivazione
	if (_a == RELU)
		Kernel::actReluK(1, _alignedNodes, output, _nodes);
	else if (_a == SIGMOID)
		Kernel::actSigmoidK(1, _alignedNodes, output, _nodes);
	else if (_a == TANH)
		Kernel::actTanhK(1, _alignedNodes, output, _nodes);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
}

void FullyConnected::back_propagation() {

}

void FullyConnected::back_propagation_output(const double *prev, const uint8_t *labels, const int &target, const double &learningRate) {
    
    // Calcolo dell'errore per ogni nodo
    Kernel::outputErrorK(1, _alignedNodes, output, error, labels, target, _nodes);
    
    // CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
    
    // Applicare derivata della funzione di attivazione
	if (_a == RELU)
		Kernel::derivActReluK(1, _alignedNodes, output, error, _nodes);
	else if (_a == SIGMOID)
		Kernel::derivActSigmoidK(1, _alignedNodes, output, error, _nodes);
	else if (_a == TANH)
		Kernel::derivActTanhK(1, _alignedNodes, output, error, _nodes);
	
	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
    
    // Aggiornare i pesi (da mettere in funzione)    
    updateWeights(prev, learningRate);   

}

void FullyConnected::updateWeights(const double *prev, const double &learningRate) {
    
    // Matrice temporanea
    double *temp;
    
    // Dimensione matrice temporanea in byte
	const unsigned int wBytes = _wDim * sizeof(double);

	// Allocare la matrice temporanea
	CHECK(hipMalloc((void**)&temp, wBytes));
	
	// Riempirla di 0
	CHECK(hipMemset(temp, 0, wBytes));
	
	for (int i = 0; i < _nodes; i++)
	    CHECK_CUBLAS(
		    hipblasDaxpy(handle, _prevLayerDim, &output[i], prev, 1, temp + i, 1));
    
    // CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());
    
    // Aggiornamento effettivo dei pesi 
    CHECK_CUBLAS(
		hipblasDaxpy(handle, _wDim, &learningRate, temp, 1, weight, 1));
		
    // CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());
	
	// Aggiornamento del bias 
    CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &learningRate, error, 1, bias, 1));
		
    // CPU deve attendere che esecuzione della funzione finisca
    CHECK(hipDeviceSynchronize());
    
    // Distrugge la matrice dei pesi
	CHECK(hipFree(temp));

}

void FullyConnected::deleteCuda() {

	CHECK_CUBLAS(hipblasDestroy(handle));
	CHECK(hipFree(weight));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
}
