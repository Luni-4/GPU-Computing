#include "hip/hip_runtime.h"
#include "Kernel.h"

__global__ void initWeight(double *weight, const int wDim, hiprandState *states) {

	// Gestione degli indici	
	const unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned int tid = blockId * blockDim.x + threadIdx.x;

	// Sequenza di rand diversa per ogni thread
	hiprand_init(tid, 0, 0, &states[tid]);

	// Variabile che conterr� il valore casuale
	double r = hiprand_uniform_double(&states[tid]);

	if (tid % 2 == 0)
		r = -r;

	if (tid < wDim)
#ifdef TOYINPUT
		weight[tid] = tid;
#else
		weight[tid] = 0.4f * r;
#endif
}

void Kernel::initWeightK(dim3 b, dim3 t, double *weight, const int &wDim, hiprandState *states) {
#ifdef _WIN32
	initWeight NvCUDA2(b, t) (weight, wDim, states);
#else
	initWeight << <b, t >> > (weight, wDim, states);
#endif
}

__global__ void initBias(double *bias, const int node, hiprandState *states) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Sequenza di rand diversa per ogni thread
	hiprand_init(tid, 0, 0, &states[tid]);

	// Variabile che conterr� il valore casuale
	double r = hiprand_uniform_double(&states[tid]);

	if (tid % 2 == 0)
		r = -r;

	if (tid < node)
#ifdef TOYINPUT
		bias[tid] = 1.0f;
#else
		bias[tid] = r;
#endif
}

void Kernel::initBiasK(dim3 b, dim3 t, double *bias, const int &wDim, hiprandState *states) {
#ifdef _WIN32
	initBias NvCUDA2(b, t) (bias, wDim, states);
#else
	initBias << <b, t >> > (bias, wDim, states);
#endif
}

__global__ void outputError(const double *output, double *error, const uint8_t *label, const int target, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int trueLabel = 0;

	/* Il predittore dovrebbe predire con probabilit� 1 solo la label passata alla funzione, quindi la variabile
	trueLabel contiene il valore che ci si aspetterebbe dal predittore, cio� 1 */
	if (tid == label[target])
		trueLabel = 1;

	// L'errore commesso � dato dalla differenza tra la predizione ottenuta e il valore reale dell'etichetta
	if (tid < node)
		error[tid] = trueLabel - output[tid];
}

void Kernel::outputErrorK(dim3 b, dim3 t, const double *output, double *error, const uint8_t *label, const int &target, const int &nodes) {
#ifdef _WIN32
	    outputError NvCUDA2(b, t) (output, error, label, target, nodes);
#else
	    outputError << <b, t >> > (output, error, label, target, nodes);
#endif
}




/* Funzione di attivazione del Sigmoide e derivata */

__global__ void actRelu(double *output, double *temp, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node) {
		temp[tid] = output[tid];
		output[tid] = log(1 + exp((output[tid])));
	}
}

__global__ void derivActRelu(double *error, double *temp, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		error[tid] = error[tid] * (1 / (1 + (exp((-temp[tid])))));
}

void Kernel::actReluK(dim3 b, dim3 t, double *output, double *temp, const int &nodes) {
#ifdef _WIN32
	    actRelu NvCUDA2(b, t) (output, temp, nodes);
#else
	    actRelu << <b, t >> > (output, temp, nodes);    
#endif
}

void Kernel::derivActReluK(dim3 b, dim3 t, double *error, double *temp, const int &nodes) {
#ifdef _WIN32
	    derivActRelu NvCUDA2(b, t) (error, temp, nodes);
#else
	    derivActRelu << <b, t >> > (error, temp, nodes);
#endif 
}



/* Funzione di attivazione del Sigmoide e derivata */

__global__ void actSigmoid(double *output, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		output[tid] = 1 / (1 + (exp((-output[tid])) ));
}

__global__ void derivActSigmoid(const double *output, double *error, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	double r;

	if (tid < node) {
	    r = output[tid] * (1 - output[tid]);
		error[tid] = error[tid] * r; 
	}
}

void Kernel::actSigmoidK(dim3 b, dim3 t, double *output, const int &nodes) {
#ifdef _WIN32
	    actSigmoid NvCUDA2(b, t) (output, nodes);
#else
	    actSigmoid << <b, t >> > (output, nodes);
#endif
}

void Kernel::derivActSigmoidK(dim3 b, dim3 t, const double *output, double *error, const int &nodes) {
#ifdef _WIN32
	    derivActSigmoid NvCUDA2(b, t) (output, error, nodes);
#else
	    derivActSigmoid << <b, t >> > (output, error, nodes);
#endif
}


/* Funzione di attivazione della Tanh e derivata */

__global__ void actTanh(double *output, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		output[tid] = tanh(output[tid]);
}

__global__ void derivActTanh(const double *output, double *error, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		error[tid] = error[tid] * (1 - pow(output[tid], 2));
}

void Kernel::actTanhK(dim3 b, dim3 t, double *output, const int &nodes) {
#ifdef _WIN32
	    actTanh NvCUDA2(b, t) (output, nodes);
#else
	    actTanh << <b, t >> > (output, nodes);
#endif
}

void Kernel::derivActTanhK(dim3 b, dim3 t, const double *output, double *error, const int &nodes) {
#ifdef _WIN32
	derivActTanh NvCUDA2(b, t) (output, error, nodes);
#else
	derivActTanh << <b, t >> > (output, error, nodes);
#endif
}



__global__ void errorPrevOutput(double *temp, const double *prevOutput, const double *error, const int node, const int prevDim) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	const unsigned int column = tid % prevDim;
	const unsigned int row = (tid - column) / prevDim;	

	if (tid < node)
		temp[tid] = error[row] * prevOutput[column];
}


void Kernel::errorPrevOutputK(dim3 b, dim3 t, double *temp, const double *prevOutput, const double *error, const int &nodes, const int &dim, const int &prevDim) {
#ifdef _WIN32
	    errorPrevOutput NvCUDA2(b, t) (temp, prevOutput, error, dim, prevDim);
#else
        errorPrevOutput << <b, t >> > (temp, prevOutput, error, dim, prevDim);
#endif
}
