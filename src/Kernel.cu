#include "hip/hip_runtime.h"
#include "Kernel.h"

__global__ void initWeight(double *weight, const int wDim, hiprandState *states) {

	// Gestione degli indici	
	const unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned int tid = blockId * blockDim.x + threadIdx.x;

	// Sequenza di rand diversa per ogni thread
	hiprand_init(tid, 0, 0, &states[tid]);

	// Variabile che conterr� il valore casuale
	double r = hiprand_uniform_double(&states[tid]);

	if (tid % 2 == 0)
		r = -r;

	if (tid < wDim)
#ifdef TOYINPUT
		weight[tid] = 1.0;
#else
		weight[tid] = 0.4 * r;
#endif
}

void Kernel::initWeightK(dim3 b, dim3 t, double *weight, const int &wDim, hiprandState *states) {
#ifdef _WIN32
	initWeight NvCUDA2(b, t) (weight, wDim, states);
#else
	initWeight << <b, t >> > (weight, wDim, states);
#endif
}

__global__ void initBias(double *bias, const int node, hiprandState *states) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Sequenza di rand diversa per ogni thread
	hiprand_init(tid, 0, 0, &states[tid]);

	// Variabile che conterr� il valore casuale
	double r = hiprand_uniform_double(&states[tid]);

	if (tid % 2 == 0)
		r = -r;

	if (tid < node)
#ifdef TOYINPUT
		bias[tid] = 1.0;
#else
		bias[tid] = 0.4 * r;
#endif
}

void Kernel::initBiasK(dim3 b, dim3 t, double *weight, const int &wDim, hiprandState *states) {
#ifdef _WIN32
	initBias NvCUDA2(b, t) (weight, wDim, states);
#else
	initBias << <b, t >> > (weight, wDim, states);
#endif
}

__global__ void outputError(const double *output, double *error, const uint8_t *label, const int target, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int trueLabel = 0;

	/* Il predittore dovrebbe predire con probabilit� 1 solo la label passata alla funzione, quindi la variabile
	trueLabel contiene il valore che ci si aspetterebbe dal predittore, cio� 1 */
	if (tid == label[target])
		trueLabel = 1;

	// L'errore commesso � dato dalla differenza tra la predizione ottenuta e il valore reale dell'etichetta
	if (tid < node)
		error[tid] = trueLabel - output[tid];
}

void Kernel::outputErrorK(dim3 b, dim3 t, const double *output, double *error, const uint8_t *label, const int &target, const int &nodes) {
#ifdef _WIN32
	outputError NvCUDA2(b, t) (output, error, label, target, nodes);
#else
	outputError << <b, t >> > (output, error, label, target, nodes);
#endif
}




/* Funzione di attivazione del Sigmoide e derivata */

__global__ void actRelu(double *output, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		output[tid] = log(1 + exp((output[tid])));
}

__global__ void derivActRelu(const double *output, double *error, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		error[tid] = error[tid] * (1 / (1 + (exp((-output[tid])))));
}

void Kernel::actReluK(dim3 b, dim3 t, double *output, const int &nodes) {
#ifdef _WIN32
	actRelu NvCUDA2(b, t) (output, nodes);
#else
	actRelu << <b, t >> > (output, nodes);
#endif 
}

void Kernel::derivActReluK(dim3 b, dim3 t, const double *output, double *error, const int &nodes) {
#ifdef _WIN32
	derivActRelu NvCUDA2(b, t) (output, error, nodes);
#else
	derivActRelu << <b, t >> > (output, error, nodes);
#endif 
}



/* Funzione di attivazione del Sigmoide e derivata */

__global__ void actSigmoid(double *output, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		output[tid] = 1 / (1 + (exp((-output[tid]))));
}

__global__ void derivActSigmoid(const double *output, double *error, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		error[tid] = error[tid] * (output[tid] * (1 - output[tid]));
}

void Kernel::actSigmoidK(dim3 b, dim3 t, double *output, const int &nodes) {
#ifdef _WIN32
	actSigmoid NvCUDA2(b, t) (output, nodes);
#else
	actSigmoid << <b, t >> > (output, nodes);
#endif 
}

void Kernel::derivActSigmoidK(dim3 b, dim3 t, const double *output, double *error, const int &nodes) {
#ifdef _WIN32
	derivActSigmoid NvCUDA2(b, t) (output, error, nodes);
#else
	derivActSigmoid << <b, t >> > (output, error, nodes);
#endif 
}




/* Funzione di attivazione della Tanh e derivata */

__global__ void actTanh(double *output, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		output[tid] = tanh(output[tid]);
}

__global__ void derivActTanh(const double *output, double *error, const int node) {

	// Gestione degli indici	
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < node)
		error[tid] = error[tid] * (1 - pow(tanh(output[tid]), 2));
}

void Kernel::actTanhK(dim3 b, dim3 t, double *output, const int &nodes) {
#ifdef _WIN32
	actTanh NvCUDA2(b, t) (output, nodes);
#else
	actTanh << <b, t >> > (output, nodes);
#endif 
}

void Kernel::derivActTanhK(dim3 b, dim3 t, const double *output, double *error, const int &nodes) {
#ifdef _WIN32
	derivActTanh NvCUDA2(b, t) (output, error, nodes);
#else
	derivActTanh << <b, t >> > (output, error, nodes);
#endif 
}
