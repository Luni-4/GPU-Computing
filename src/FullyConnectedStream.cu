#include <iostream>

#ifdef _WIN32
#include "Windows.h"
#endif

// Funzioni comuni
#include "Common.h"

// Cuda Kernel
#include "KernelStreamCPU.h"

// Classi
#include "FullyConnectedStream.h"

FullyConnectedStream::FullyConnectedStream(const int &width, const int &height, const ActFctType &a)
	: LayerDefinition(width, height, 1, FULLY_CONNECTED, a) {

	this->_nodes = width * height;

	initStreams();

}

FullyConnectedStream::FullyConnectedStream(const int &width, const ActFctType &a)
	: LayerDefinition(width, 1, 1, FULLY_CONNECTED, a),
	_nodes(width) {

	initStreams();

}

FullyConnectedStream::~FullyConnectedStream() {

}


std::vector<double> FullyConnectedStream::getWeights(void) {
	std::vector<double> wCPU(_wDim);
	CHECK(hipMemcpy(&wCPU[0], weight, _wBytes, hipMemcpyDeviceToHost));
	return wCPU;
}

std::vector<double> FullyConnectedStream::getBias(void) {
	std::vector<double> bCPU(_nodes);
	CHECK(hipMemcpy(&bCPU[0], bias, _nodes * sizeof(double), hipMemcpyDeviceToHost));
	return bCPU;
}

int FullyConnectedStream::getPredictionIndex(void) {
	int maxIndex;

	// Individuare indice (classe) che corrisponde al valore massimo di output
	CHECK_CUBLAS(
		hipblasIdamax(handle, _nodes, output, 1, &maxIndex));

	return maxIndex - 1;
}

void FullyConnectedStream::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {

	// Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));

	// Impostazioni della cache
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	// Creazione degli stream
	streams = (hipStream_t *)malloc(_nStreams * sizeof(hipStream_t));

	for (int i = 0; i < _nStreams; i++) {
		CHECK(hipStreamCreate(&(streams[i])));
	}

	// Dimensione matrice dei pesi
	_wDim = prevLayerWidth * prevLayerHeight * prevLayerDepth * _nodes;

	// Salvare dimensione del livello precedente
	_prevLayerDim = prevLayerWidth * prevLayerHeight * prevLayerDepth;

	// Dimensione matrice dei pesi in byte
	_wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);

#ifdef DEBUG
	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);
#endif

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, _wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	CHECK(hipMalloc((void**)&error, Bytes));
	CHECK(hipMalloc((void**)&temp, _wBytes));

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(prevLayerWidth * prevLayerHeight, THREADS);

	// Tanti blocchi quanto sono i nodi e la profondità del layer precedente
	dim3 numBlocks(_nodes, prevLayerDepth, 1);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi del livello precedente
	dim3 threadBlocks(aligned, 1, 1);

	// Inizializza array per numeri casuali
	hiprandStateXORWOW_t *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _nodes * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandStateXORWOW_t)));

	// Inizializzare i weight del livello
	KernelStream::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// Inizializzare i bias del livello
	KernelStream::initBiasK(_alignedNodes / THREADS, THREADS, bias, _nodes, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore dei pesi\n\n";
	pettyPrintCuda(weight, _wDim, _prevLayerDim);
	std::cout << "\n\nValore dei bias\n\n";
	pettyPrintCuda(bias, _nodes, 1);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}


void FullyConnectedStream::forward_propagation(const double *prevOutput) {

	for (int i = 0; i < _nStreams; i++) {
		int indexW = i * _alignedMatrix * _prevLayerDim;
		int indexO = i * _alignedMatrix;
		CHECK_CUBLAS(hipblasSetStream(handle, streams[i]));
		CHECK_CUBLAS(
			hipblasDgemv(handle, HIPBLAS_OP_T, _prevLayerDim, _alignedMatrix, &alpha, weight + indexW, _prevLayerDim, prevOutput, 1, &beta, output + indexO, 1));
	}

#ifdef DEBUG
	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nOutput dei nodi senza bias\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif

	for (int i = 0; i < _nStreams; i++) {
		int indexO = i * _alignedMatrix;
		CHECK_CUBLAS(hipblasSetStream(handle, streams[i]));
		CHECK_CUBLAS(
			hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _alignedMatrix, &alpha, bias + indexO, 1, &alpha, output + indexO, 1, output + indexO, 1));
	}

#ifdef DEBUG
	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nOutput dei nodi con bias sommato\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif    

	// Applicare funzione di attivazione
	if (_a == RELU)
		KernelStream::actReluK(1, _alignedMatrix, streams, _nStreams, output, temp, _nodes);
	else if (_a == SIGMOID)
		KernelStream::actSigmoidK(1, _alignedMatrix, streams, _nStreams, output, _nodes);
	else if (_a == TANH)
		KernelStream::actTanhK(1, _alignedMatrix, streams, _nStreams, output, _nodes);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nOutput dei nodi con funzione di attivazione\n\n";
	pettyPrintCuda(output, _nodes, 1);
#endif
}

void FullyConnectedStream::back_propagation_output(const double *prevOutput, const uint8_t *labels, const int &target, const double &learningRate) {

	// Applicare derivata della funzione di attivazione
	if (_a == RELU)
		KernelStream::derivActReluK(1, _alignedMatrix, streams, _nStreams, error, temp, _nodes);
	else if (_a == SIGMOID)
		KernelStream::derivActSigmoidK(1, _alignedMatrix, streams, _nStreams, output, error, _nodes);
	else if (_a == TANH)
		KernelStream::derivActTanhK(1, _alignedMatrix, streams, _nStreams, output, error, _nodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi con relativa derivata\n\n";
	pettyPrintCuda(error, _nodes, 1);
#endif

	// Calcolo dell'errore per ogni nodo
	KernelStream::outputErrorK(_alignedNodes / THREADS, THREADS, output, error, labels, target, _nodes);

	//CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nErrore commesso sui nodi back propagation output\n\n";
	pettyPrintCuda(error, _nodes, 1);
#endif

	int m = ALIGN_UP(_prevLayerDim, _nStreams) / _nStreams;

	// Propagazione dell'errore dal livello successivo
	for (int i = 0; i < _nStreams; i++) {
		int indexW = i * m * _nodes;
		int indexO = i * m;
		CHECK_CUBLAS(hipblasSetStream(handle, streams[i]));
		CHECK_CUBLAS(
			hipblasDgemv(handle, HIPBLAS_OP_N, m, _nodes, &alpha, weight + indexW, m, error, 1, &beta, prevError + indexO, 1));
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nForward weight\n\n";
	pettyPrintCuda(weight, _wDim, prevNodes);
	std::cout << "\n\nForward error\n\n";
	pettyPrintCuda(error, _nodes, 1);
	std::cout << "\n\nErrore commesso sui nodi back propagation\n\n";
	pettyPrintCuda(prevError, prevNodes, 1);
#endif

	// Aggiornare i pesi (da mettere in funzione)    
	updateWeights(prevOutput, learningRate);
}

void FullyConnectedStream::back_propagation(const double *prevOutput, double *prevErr, const double &learningRate, const bool notFirst) {

	// Applicare derivata della funzione di attivazione
	if (_a == RELU)
		KernelStream::derivActReluK(1, _alignedMatrix, streams, _nStreams, error, temp, _nodes);
	else if (_a == SIGMOID)
		KernelStream::derivActSigmoidK(1, _alignedMatrix, streams, _nStreams, output, error, _nodes);
	else if (_a == TANH)
		KernelStream::derivActTanhK(1, _alignedMatrix, streams, _nStreams, output, error, _nodes);

	// Prodotto prevErr * error
	KernelStream::prevErrorK(_alignedNodes / THREADS, THREADS, prevErr, error, _nodes);

	// Calcolo del nuovo prevError
	CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_N, _prevLayerDim, _nodes, &alpha, weight, _prevLayerDim, error, 1, &beta, prevError, 1));

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi con relativa derivata\n\n";
	pettyPrintCuda(error, _nodes, 1);
#endif

	// Aggiornare i pesi (da mettere in funzione)    
	updateWeights(prevOutput, learningRate);
}

void FullyConnectedStream::updateWeights(const double *prevOutput, const double &learningRate) {

	int dim = ALIGN_UP(_alignedMatrix * _prevLayerDim, THREADS);

	KernelStream::errorPrevOutputK(dim / THREADS, THREADS, streams, _nStreams, temp, prevOutput, error, _alignedMatrix, _alignedMatrix * _prevLayerDim, _prevLayerDim);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice temporanea per aggiornamento pesi\n\n";
	pettyPrintCuda(temp, _wDim, _prevLayerDim);
#endif

	for (int i = 0; i < _nStreams; i++) {
		int indexW = i * _alignedMatrix * _prevLayerDim;
		CHECK_CUBLAS(hipblasSetStream(handle, streams[i]));
		CHECK_CUBLAS(
			hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, _alignedMatrix, _prevLayerDim, &learningRate, temp + indexW, _alignedMatrix, &alpha, weight + indexW, _alignedMatrix, weight + indexW, _alignedMatrix));
	}

	//CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice dei pesi aggiornata\n\n";
	pettyPrintCuda(weight, _wDim, _prevLayerDim);
#endif

	for (int i = 0; i < _nStreams; i++) {
		int indexO = i * _alignedMatrix;
		CHECK_CUBLAS(hipblasSetStream(handle, streams[i]));
		CHECK_CUBLAS(
			hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _alignedMatrix, &learningRate, error + indexO, 1, &alpha, bias + indexO, 1, bias + indexO, 1));
	}

	//CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nVettore del bias aggiornato\n\n";
	pettyPrintCuda(bias, _nodes, 1);
#endif
}

inline void FullyConnectedStream::initStreams(void) {

	this->_alignedNodes = ALIGN_UP(_nodes, THREADS);

	// Numero degli stream
	this->_nStreams = 2;

	// Numero di elementi che uno stream deve elaborare
	this->_alignedMatrix = _nodes / _nStreams;
}

void FullyConnectedStream::deleteCuda(void) {

	CHECK_CUBLAS(hipblasDestroy(handle));

	for (int i = 0; i < _nStreams; i++) {
		CHECK(hipStreamDestroy(streams[i]));
	}

	CHECK(hipFree(weight));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
	CHECK(hipFree(temp));

	free(streams);
}

void FullyConnectedStream::printW() {
	//printFromCudaFormatted(weight, _wDim, _prevLayerDim);
}
