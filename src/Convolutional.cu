#include "hip/hip_runtime.h"
#ifdef _WIN32
#include "Windows.h"
#endif

#include <iostream>
#include "Common.h"

#define DEFAULT

// Cuda Kernel
#include "KernelCPU.h"

#include "Convolutional.h"

Convolutional::Convolutional(const int &filterWidth, const int &depth, const int &stride, const ActFctType &a)
	: LayerDefinition(0, 0, depth, CONVOLUTIONAL, a) {
	this->_filterWidth = filterWidth;
	this->_filterDim = filterWidth * filterWidth;
	this->_depth = depth;
	this->_stride = stride;
	this->_padding = 0;
}

Convolutional::~Convolutional() {
}

std::vector<double> Convolutional::getWeights(void) {
	std::vector<double> wCPU(_wDim);
	CHECK(hipMemcpy(&wCPU[0], weight, _wBytes, hipMemcpyDeviceToHost));
	return wCPU;
}

std::vector<double> Convolutional::getBias(void) {
	std::vector<double> bCPU(_nodes);
	CHECK(hipMemcpy(&bCPU[0], bias, _nodes * sizeof(double), hipMemcpyDeviceToHost));
	return bCPU;
}

int Convolutional::getPredictionIndex(void) {
	int maxIndex;

	// Individuare indice (classe) che corrisponde al valore massimo di output
	CHECK_CUBLAS(
		hipblasIdamax(handle, _nodes, output, 1, &maxIndex));

	return maxIndex - 1;
}

void Convolutional::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {

	_prevLayerWidth = prevLayerWidth;
	_prevLayerDepth = prevLayerDepth;

	//numero di nodi dipende da filtro e nodi livello precedente
	//width
	_width = _calcOutput(false);
	//height
	_height = _calcOutput(false);
	//depth = numero di filtri

	this->_nodes = _width * _height * _depth;
	_alignedNodes = ALIGN_UP(_nodes, THREADS);

	_uniqueNodes = _width * _height;

	streams = (hipStream_t *)malloc(_uniqueNodes * sizeof(hipStream_t));

	for (int i = 0; i < _uniqueNodes; i++) {
		CHECK(hipStreamCreate(&(streams[i])));
	}

#ifdef DEBUG
	std::cout << "dimensioni output del livello: " << _width << " - " << _height << " - " << _depth << std::endl;
#endif

	//Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));

	// Impostazioni della cache
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	// Dimensione matrice dei pesi
	_wDim = _filterDim * prevLayerDepth * _depth;

	// Dimensione matrice dei pesi in byte
	_wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);

#ifdef DEBUG
	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);
#endif

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, _wBytes));
	CHECK(hipMalloc((void**)&weightRot, _wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	//CHECK(hipMalloc((void**)&output, (_width*2) * (_width*2) * (_uniqueNodes / (_width *2)) * sizeof(double)));
	CHECK(hipMalloc((void**)&error, Bytes));
	CHECK(hipMalloc((void**)&errorRot, Bytes));
	CHECK(hipMalloc((void**)&tempWeight, _wBytes));
	CHECK(hipMalloc((void**)&tempOutput, Bytes));

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo che compone un blocco di output * la profondit� del livello precedente e grande quanto un filtro
	unsigned int subBytes = _uniqueNodes * _prevLayerDepth * _filterDim * sizeof(double);
	CHECK(hipMalloc((void**)&subForward, subBytes));

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di output di L-1
	const int prevUniqueNodes = _prevLayerWidth * _prevLayerWidth;
	subBytes = prevUniqueNodes * _depth * _filterDim * sizeof(double);
	CHECK(hipMalloc((void**)&subCalcError, subBytes));

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di filtro
	//(prima genero sottomatrici grandi quanto _filterDim e ne genero tante quante uniqueNodes,
	// ora genero sottomatrici grandi quanto uniqueNodes e ne genero tante quante _filterDim)
	subBytes = _uniqueNodes * _prevLayerDepth * _filterDim * sizeof(double);
	CHECK(hipMalloc((void**)&subBack, subBytes));

	// matrice temporanea inizializzata a 0 per zero padding
	paddingWidth = (_filterWidth - 1) * 2 + _width;
	const int uniquePadding = paddingWidth * paddingWidth;
	paddingSize = uniquePadding * _depth; //come output 
	CHECK(hipMalloc((void**)&padding, paddingSize * sizeof(double)));
	CHECK(hipMemset(padding, 0, paddingSize * sizeof(double)));

#ifdef DEBUG
	std::cout << "Memoria allocata \n" << std::endl;
#endif

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(_filterDim, THREADS);

	// Tanti blocchi quanto sono i filtri e la profondit� del layer precedente
	dim3 numBlocks(_depth, prevLayerDepth, 1);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi che compongono i filtri
	dim3 threadBlocks(aligned, 1, 1);

	// Inizializza array per numeri casuali
	hiprandStateXORWOW_t *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _nodes * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandStateXORWOW_t)));

	// Inizializzare i weight del livello
	Kernel::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// Inizializzare i bias del livello
	Kernel::initBiasK((_alignedNodes / THREADS), THREADS, bias, _nodes, devStates);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore dei pesi\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
	std::cout << "\n\nValore dei bias\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
	std::cout << "\n\n\n\n";
#endif

	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	threadBlocks = dim3(_depth, _prevLayerDepth, 1);

	// Tanti blocchi quante sono le righe e le colonne di forwardError
	numBlocks = dim3(_filterWidth, _filterWidth, 1);

	Kernel::rot180BisK(numBlocks, threadBlocks, weight, weightRot, _filterDim);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore dei pesi ruotati\n\n";
	printFromCudaFormatted(weightRot, _wDim, _filterWidth);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}

void Convolutional::forward_propagation(const double * prevOutput) {

#ifdef DEBUG
	std::cout << "\n\nValore dell'input\n\n";
	printFromCudaFormatted(prevOutput, _prevLayerWidth * _prevLayerWidth * _prevLayerDepth, _prevLayerWidth);
#endif

#ifdef DEFAULT

    int p = _width * 4;

	// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
	dim3 threadBlocks(_filterWidth, _filterWidth, 1);

	// Tanti blocchi quanti sono i nodi in output e il depth del livello precedente
	dim3 numBlocks(_width, _height, _prevLayerDepth);

	Kernel::createSubmatrixBisK(numBlocks, threadBlocks, subForward, prevOutput, _prevLayerWidth, _filterWidth, _stride, _uniqueNodes);

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici\n\n";
	printFromCudaFormatted(subForward, _uniqueNodes * _prevLayerDepth * _filterDim, _filterWidth);
#endif

	//ora sono in una situazione simile al fully connected
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {

			CHECK_CUBLAS(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _uniqueNodes, _filterDim, &alpha, weightRot + (i * _filterDim * _prevLayerDepth) + (j * _filterDim), 1, subForward + (j * _uniqueNodes), _filterDim, &beta, output + (i * _uniqueNodes), 1));
			//CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _filterDim, _uniqueNodes, &alpha, subForward + (j * _uniqueNodes), _filterDim, weightRot + (i * _filterDim * _prevLayerDepth) + (j * _filterDim), 1, &beta, output + (i * _uniqueNodes), 1));
            
            // Stream su prodotto tra matrici
			/*for (int nS = 0; nS < (_uniqueNodes / p); nS++) {
				int subForwardPlus = p * _filterDim * nS;
				int o = p * p * nS;
				CHECK_CUBLAS(hipblasSetStream(handle, streams[nS]));
				CHECK_CUBLAS(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, p, p, _filterDim, &alpha, (subForward + subForwardPlus), p, (subForward + subForwardPlus), _filterDim, &beta, output + o, p));
			}*/
			
			/*for (int nS = 0; nS < _uniqueNodes / p; nS++) {
				int subForwardPlus = p * _filterDim * nS;
				int o = p * nS;
				CHECK_CUBLAS(hipblasSetStream(handle, streams[nS]));
				CHECK_CUBLAS(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, p, _filterDim, &alpha, weightRot, 1, subForward + subForwardPlus, _filterDim, &beta, output + o , 1));
			}*/			
		}
	}
#else
	// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
	dim3 threadBlocks(_filterWidth, _filterWidth, 1);

	// Tanti blocchi quanti sono i nodi in output e il depth del livello precedente
	dim3 numBlocks(_width, _height, _prevLayerDepth);

	Kernel::createSubmatrixProductK(numBlocks, threadBlocks, subForward, prevOutput, weightRot, _prevLayerWidth, _filterWidth, _stride, _uniqueNodes);

	CHECK(hipDeviceSynchronize());

	// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
	threadBlocks = dim3(_width, _height, 1);

	// Tanti blocchi quanti sono i nodi in output e il depth del livello precedente
	numBlocks = dim3(1, 1, 1);

	//Kernel::outputFromSubK(numBlocks, threadBlocks, output, subForward, _filterDim);

	//for (int i = 0; i < _depth; i++) {
	//	for (int j = 0; j < _prevLayerDepth; j++) {
	//		for (int nS = 0; nS < _uniqueNodes; nS++) {
	//			int subForwardPlus = _filterDim * nS;
	//			CHECK_CUBLAS(hipblasSetStream(handle, streams[nS]));
	//			CHECK_CUBLAS(hipblasDasum(handle, _filterDim, subForward + (j * _uniqueNodes) + subForwardPlus, 1, output + (i * _uniqueNodes) + nS));
	//		}
	//	}
	//}
#endif

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore output senza bias\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Somma con il bias
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &alpha, bias, 1, output, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _nodes, &alpha, bias, 1, &alpha, output, 1, output, 1));

#ifdef DEBUG
	std::cout << "\n\nValore output prima di funzione di attivazione\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Applicare funzione di attivazione
	if (_a == RELU)
		Kernel::actReluK((_alignedNodes / THREADS), THREADS, output, tempOutput, _nodes);
	else if (_a == SIGMOID)
		Kernel::actSigmoidK((_alignedNodes / THREADS), THREADS, output, _nodes);
	else if (_a == TANH)
		Kernel::actTanhK((_alignedNodes / THREADS), THREADS, output, _nodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore output *************************************************\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

}

void Convolutional::calcError(double *prevError, const int &prevNodes) {

	//prev error � l'errore del livello precedente che devo riempire, 
	//error � l'errore che ho usato al passo precedente (non ruotato) quando sono passato da questo livello

#ifdef DEBUG
	std::cout << "\n\error in calc error\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	// Blocchi bidimensionali contenenti tanti thread quanti sono i nodi in output
	dim3 threadBlocks(_height, _width, 1);

	// Tanti blocchi quanto il numero di filtri
	dim3 numBlocks(1, 1, _depth);

	Kernel::zeroPaddingBisK(numBlocks, threadBlocks, padding, error, _width, _filterWidth);

#ifdef DEBUG
	std::cout << "\n\nerror con zero padding\n\n";
	printFromCudaFormatted(padding, paddingSize, paddingWidth);
#endif

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di output di L-1
	const int prevUniqueNodes = prevNodes / _prevLayerDepth;

	// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
	threadBlocks = dim3(_filterWidth, _filterWidth, 1);

	// Tanti blocchi quanti sono i nodi in input e il depth del livello precedente
	numBlocks = dim3(sqrt(prevUniqueNodes), sqrt(prevUniqueNodes), _prevLayerDepth);

	Kernel::createSubmatrixBisK(numBlocks, threadBlocks, subCalcError, padding, paddingWidth, _filterWidth, _stride, prevUniqueNodes);

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici zero padding\n\n";
	printFromCudaFormatted(subCalcError, prevUniqueNodes * _depth * _filterDim, _filterWidth);
#endif

	//ora sono in una situazione simile alla convoluzione
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {
			CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _filterDim, prevUniqueNodes, &alpha, subCalcError + (i * prevUniqueNodes), _filterDim, weightRot + ((i + j * _depth) * _filterDim), 1, &beta, prevError + (j * prevUniqueNodes), 1));
		}
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi back propagation\n\n";
	printFromCudaFormatted(prevError, prevNodes, sqrt(prevUniqueNodes));
#endif

}

void Convolutional::back_propagation(const double *prevOutput, const double &learningRate) {
	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);
}

void Convolutional::back_propagation_output(const double * prevOutput, const uint8_t * labels, const int & target, const double & learningRate) {
	// Calcolo dell'errore per ogni nodo
	Kernel::outputErrorK((_alignedNodes / THREADS), THREADS, output, error, labels, target, _nodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi back propagation output\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);
}

void Convolutional::calcBackPropagation(const double *prevOutput, const double &learningRate) {

	// Blocchi bidimensionali contenenti tanti thread quanti il numero di depth in output
	dim3 threadBlocks(_depth, 1, 1);

	// Tanti blocchi quante sono le righe e le colonne di error
	dim3 numBlocks(_width, _height, 1);

	Kernel::rot180BisK(numBlocks, threadBlocks, error, errorRot, _uniqueNodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore error ruotato\n\n";
	printFromCudaFormatted(errorRot, _nodes, _width);
#endif

	// Applicare derivata della funzione di attivazione
	if (_a == RELU)
		Kernel::derivActReluK((_alignedNodes / THREADS), THREADS, errorRot, tempOutput, _nodes);
	else if (_a == SIGMOID)
		Kernel::derivActSigmoidK((_alignedNodes / THREADS), THREADS, output, errorRot, _nodes);
	else if (_a == TANH)
		Kernel::derivActTanhK((_alignedNodes / THREADS), THREADS, output, errorRot, _nodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi con relativa derivata\n\n";
	printFromCudaFormatted(errorRot, _nodes, _width);
#endif

	updateWeights(prevOutput, learningRate);
}

void Convolutional::updateWeights(const double *prevOutput, const double &learningRate) {

	// Blocchi tridimensionali contenenti tanti thread quanti sono i nodi in output
	dim3 threadBlocks(_width, _height, 1);

	// Tanti blocchi quanti la grandezza dei filtri e il depth del livello precedente
	dim3 numBlocks(_filterWidth, _filterWidth, _prevLayerDepth);

	Kernel::createSubmatrixBisK(numBlocks, threadBlocks, subBack, prevOutput, _prevLayerWidth, _width, _stride, _filterDim);

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici backpropagation\n\n";
	printFromCudaFormatted(subBack, _uniqueNodes * _filterDim, _width);
#endif

	//ora sono in una situazione simile al fully connected
	//double backAlpha = 1.0 / _uniqueNodes;
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {
			CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _uniqueNodes, _filterDim, &alpha, subBack + (j * _filterDim), _uniqueNodes, errorRot + (i * _uniqueNodes), 1, &beta, tempWeight + ((i + j * _depth) * _filterDim), 1));
		}
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice temporanea per aggiornamento pesi\n\n";
	printFromCudaFormatted(tempWeight, _wDim, _filterWidth);
#endif

	// Aggiornamento effettivo dei pesi 
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _wDim, &learningRate, tempWeight, 1, weight, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, _wDim, _depth, &learningRate, tempWeight, _wDim, &alpha, weight, _wDim, weight, _wDim));

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice dei pesi aggiornata\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
#endif

	// Ruoto subito i pesi aggiornati per poi usarli nella backpropagation al livello L-1
	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	threadBlocks = dim3(_depth, _prevLayerDepth, 1);

	// Tanti blocchi quante sono le righe e le colonne di forwardError
	numBlocks = dim3(_filterWidth, _filterWidth, 1);

	Kernel::rot180BisK(numBlocks, threadBlocks, weight, weightRot, _filterDim);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore dei pesi ruotati\n\n";
	printFromCudaFormatted(weightRot, _wDim, _filterWidth);
#endif

	// Aggiornamento del bias 
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &learningRate, errorRot, 1, bias, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _nodes, &learningRate, errorRot, 1, &alpha, bias, 1, bias, 1));

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nVettore del bias aggiornato\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
#endif

}

void Convolutional::deleteCuda() {
	CHECK_CUBLAS(hipblasDestroy(handle));
	CHECK(hipFree(weight));
	CHECK(hipFree(weightRot));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
	CHECK(hipFree(errorRot));
	CHECK(hipFree(tempWeight));
	CHECK(hipFree(tempOutput));
	CHECK(hipFree(subForward));
	CHECK(hipFree(subCalcError));
	CHECK(hipFree(subBack));
	CHECK(hipFree(padding));
}

void Convolutional::printW() {
	printFromCudaFormatted(weight, _wDim, _filterWidth);
	//printFromCudaFormatted(bias, _nodes, _width);
}

int Convolutional::_calcOutput(bool withPadding) {
	//PER ORA NON CONSIDERATO CASO IN CUI SI GENERANO ERRORI (padding numero non intero, filtro pi� grande dell'input, stride che non combacia, ecc)
	if (_filterWidth > _prevLayerWidth) {
		std::cerr << "Le dimensioni del filtro superano le dimensioni del livello precedente!!" << std::endl;
		exit(1);
	}

	if (withPadding) {
		_padding = (_filterWidth - 1) / 2;
		return _prevLayerWidth;
	}

	//+(_stride - 1)) serve per aggiornare per eccesso
	return ((_prevLayerWidth - _filterWidth + (_stride - 1)) / _stride) + 1;
}


/*
ESEMPIO DIFFERENZA UTILIZZO METODI BIS E NON:

// Blocchi bidimensionali contenenti tanti thread quanti il depth del livello precedente
dim3 threadBlocks(_prevLayerDepth, 1, 1);

// Tanti blocchi quanti sono i nodi in output (width * height), in questo modo nel kernel sfrutto gli id per righe e colonne delle submatrici
dim3 numBlocks(_width, _height, 1);

Kernel::createSubmatrixK(numBlocks, threadBlocks, sub, prevOutput, _prevLayerWidth, _filterWidth, _stride, _uniqueNodes);

**************************************

// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
dim3 threadBlocks(_filterWidth, _filterWidth, 1);

// Tanti blocchi quanti sono i nodi in output e il depth del livello precedente
dim3 numBlocks(_width, _height, _prevLayerDepth);

Kernel::createSubmatrixBisK(numBlocks, threadBlocks, sub, prevOutput, _prevLayerWidth, _filterWidth, _stride, _uniqueNodes);
*/
