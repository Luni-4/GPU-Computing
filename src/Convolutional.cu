#include "hip/hip_runtime.h"
#ifdef _WIN32
#include "Windows.h"
#endif

#include <iostream>
#include <vector>
#include <algorithm>

// Cuda Kernel
#include "Kernel.h"

#include "Common.h"
#include "Convolutional.h"

__global__ void createSubmatrix(double * sub, const double * prevOutput, const int prevLayerWidth, const int filterWidth, const int stride, const int uniqueNodes) {

	// Gestione degli indici	
	const unsigned int blockId = blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
	const unsigned int tid = blockId + threadIdx.x;

	//tid rappresenta il nodo di output
	//blockIdx.x rappresenta la colonna da cui inizia la submatrice
	//blockIdx.y rappresenta la riga da cui inizia la submatrice

	//Estraggo submatrici
	if (tid < uniqueNodes) {
		for (int i = 0; i < filterWidth; i++) {
			memcpy((sub + i * filterWidth + tid * filterWidth * filterWidth), (prevOutput + blockIdx.x * stride + (blockIdx.y * stride + i) * prevLayerWidth), filterWidth * sizeof(double));
		}
	}
}

void createSubmatrixK(dim3 b, dim3 t, double * sub, const double * prevOutput, const int prevLayerWidth, const int filterWidth, const int stride, const int uniqueNodes) {
#ifdef _WIN32
	createSubmatrix NvCUDA2(b, t) (sub, prevOutput, prevLayerWidth, filterWidth, stride, uniqueNodes);
#else
	createSubmatrix << <b, t >> > (sub, prevOutput, prevLayerWidth, filterWidth, stride, uniqueNodes);
#endif
}

__global__ void zeroPadding(double * error, const double * forwardError, const int forwardErrorWidth, const int forwardFilterWidth) {
	//blockIdx.y rappresenta la riga 
	const unsigned int p = forwardFilterWidth - 1;
	const unsigned int d = forwardErrorWidth + (p * 2);
	const unsigned int tid = ((blockIdx.y + p) * d) + p;

	memcpy((error + tid), (forwardError + blockIdx.y * forwardErrorWidth), (forwardErrorWidth * sizeof(double)));
}

void zeroPaddingK(dim3 b, dim3 t, double * error, const double * forwardError, const int forwardErrorWidth, const int forwardFilterWidth) {
#ifdef _WIN32
	zeroPadding NvCUDA2(b, t) (error, forwardError, forwardErrorWidth, forwardFilterWidth);
#else
	zeroPadding << <b, t >> > (error, forwardError, forwardErrorWidth, forwardFilterWidth);
#endif
}

Convolutional::Convolutional(const int &filterWidth, const int &depth, const int &stride, const ActFctType &a)
	: LayerDefinition(0, 0, depth, CONVOLUTIONAL, a) {
	this->_filterWidth = filterWidth;
	this->_filterDim = filterWidth * filterWidth;
	this->_depth = depth;
	this->_stride = stride;
	this->_padding = 0;
}

Convolutional::~Convolutional() {
}

// TEST
std::vector<double> Convolutional::getWeights() {
	std::vector<double> wCPU(_wDim);
	CHECK(hipMemcpy(&wCPU[0], weight, _wBytes, hipMemcpyDeviceToHost));
	return wCPU;
}

// TEST
std::vector<double> Convolutional::getBias() {
	std::vector<double> bCPU(_nodes);
	CHECK(hipMemcpy(&bCPU[0], bias, _nodes * sizeof(double), hipMemcpyDeviceToHost));
	return bCPU;
}

// TEST
uint8_t Convolutional::getPredictionIndex(void) {
	int maxIndex;

	// Individuare indice (classe) che corrisponde al valore massimo di output
	CHECK_CUBLAS(
		hipblasIdamax(handle, _nodes, output, 1, &maxIndex));

	return maxIndex;
}

void Convolutional::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {
	_prevLayerWidth = prevLayerWidth;
	_prevLayerDepth = prevLayerDepth;

	//numero di nodi dipende da filtro e nodi livello precedente
	//width
	_width = _calcOutput(false);
	//height
	_height = _calcOutput(false);
	//depth = numero di filtri

	this->_nodes = _width * _height * _depth;
	_alignedNodes = ALIGN_UP(_nodes);

#ifdef DEBUG
	std::cout << "dimensioni output del livello: " << _width << " - " << _height << " - " << _depth << std::endl;
#endif

	// Dimensione matrice dei pesi
	_wDim = _filterDim * prevLayerDepth * _depth;

	// Dimensione matrice dei pesi in byte
	_wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);

#ifdef DEBUG
	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);
#endif

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, _wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	CHECK(hipMalloc((void**)&error, Bytes));
	CHECK(hipMalloc((void**)&temp, _wBytes));

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(_filterDim);

	// Tanti blocchi quanto sono i filtri e la profondit� del layer precedente
	dim3 numBlocks(_depth, prevLayerDepth, 1);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi che compongono i filtri
	dim3 threadBlocks(aligned, 1, 1);

	// Inizializza array per numeri casuali
	hiprandState *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _nodes * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandState)));

	// Inizializzare i weight del livello
	Kernel::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

	// Inizializzare i bias del livello
	Kernel::initBiasK((_alignedNodes / THREADS), THREADS, bias, _nodes, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore dei pesi\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
	std::cout << "\n\nValore dei bias\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}

void Convolutional::forward_propagation(const double * prevOutput) {
#ifdef DEBUG
	std::cout << "\n\nValore dell'input\n\n";
	printFromCudaFormatted(prevOutput, _prevLayerWidth * _prevLayerWidth, _prevLayerWidth);
#endif

	double *sub; // Submatrici

				 // Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di output
	int uniqueNodes = _width * _height;
	const unsigned int subBytes = uniqueNodes * _filterDim * sizeof(double);

	// Alloco submatrice
	CHECK(hipMalloc((void**)&sub, subBytes));

	// Blocchi bidimensionali contenenti tanti thread quanti il depth del livello precedente
	dim3 threadBlocks(_prevLayerDepth, 1, 1);

	// Tanti blocchi quanti sono i nodi in output (width * height), in questo modo nel kernel sfrutto gli id per righe e colonne delle submatrici
	dim3 numBlocks(_width, _height, 1);

	createSubmatrixK(numBlocks, threadBlocks, sub, prevOutput, _prevLayerWidth, _filterWidth, _stride, uniqueNodes);

	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore submatrici\n\n";
	printFromCudaFormatted(sub, uniqueNodes * _filterDim, _filterWidth);
#endif

	//Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));

	//ora sono in una situazione simile al fully connected
	for (int i = 0; i < _depth; i++) {
		CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _filterDim, uniqueNodes, &alpha, sub, _filterDim, weight + (i * _filterDim), 1, &beta, output + (i * uniqueNodes), 1));
	}

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore output senza bias\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Somma con il bias
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &alpha, bias, 1, output, 1));

#ifdef DEBUG
	std::cout << "\n\nValore output prima di funzione di attivazione\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Applicare funzione di attivazione
	if (_a == RELU)
		Kernel::actReluK((_alignedNodes / THREADS), THREADS, output, _nodes);
	else if (_a == SIGMOID)
		Kernel::actSigmoidK((_alignedNodes / THREADS), THREADS, output, _nodes);
	else if (_a == TANH)
		Kernel::actTanhK((_alignedNodes / THREADS), THREADS, output, _nodes);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore output\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	CHECK(hipFree(sub));
}

void Convolutional::back_propagation(const double *prevOutput, const double *forwardWeight, const double *forwardError, const int &forwardNodes, const double &learningRate) {

	const int forwardErrorWidth = 20;
	const int forwardFilterWidth = 5;
	const int forwardStride = 1;
	const int forwardDepth = 1;

#ifdef DEBUG
	std::cout << "\n\nForward error\n\n";
	printFromCudaFormatted(forwardError, forwardErrorWidth * forwardErrorWidth, forwardErrorWidth);
#endif

	// matrice temporanea inizializzata a 0 per zero padding
	double *padding;
	const int pBytes = (forwardFilterWidth - 1) * 2 + forwardErrorWidth;
	CHECK(hipMalloc((void**)&padding, pBytes * pBytes * sizeof(double)));
	CHECK(hipMemset(padding, 0, pBytes * pBytes * sizeof(double)));

	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	dim3 threadBlocks(_depth, 1, 1);//????????????????????????????

	// Tanti blocchi quante sono le righe di forwardError, in questo modo nel kernel sfrutto gli id.y per righe
	dim3 numBlocks(1, forwardErrorWidth, 1);

	zeroPaddingK(numBlocks, threadBlocks, padding, forwardError, forwardErrorWidth, forwardFilterWidth);

#ifdef DEBUG
	std::cout << "\n\nForward error con zero padding\n\n";
	printFromCudaFormatted(padding, pBytes * pBytes, pBytes);
#endif

	double *sub; // Submatrici

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di output
	const int uniqueNodes = _width * _height;
	const int forwardFilterDim = forwardFilterWidth * forwardFilterWidth;
	const unsigned int subBytes = uniqueNodes * forwardFilterDim * sizeof(double);

	// Alloco submatrice
	CHECK(hipMalloc((void**)&sub, subBytes));

	// Blocchi bidimensionali contenenti tanti thread quanti il depth del livello precedente
	threadBlocks = dim3(_prevLayerDepth, 1, 1);

	// Tanti blocchi quanti sono i nodi in output (width * height), in questo modo nel kernel sfrutto gli id per righe e colonne delle submatrici
	numBlocks = dim3(_width, _height, 1);

	createSubmatrixK(numBlocks, threadBlocks, sub, padding, pBytes, forwardFilterWidth, forwardStride, uniqueNodes);

	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore submatrici zero padding\n\n";
	printFromCudaFormatted(sub, uniqueNodes * forwardFilterDim, forwardFilterWidth);
#endif

	//ora sono in una situazione simile alla convoluzione
	for (int i = 0; i < _depth; i++) { //?????????????????????????????
		CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, forwardFilterDim, uniqueNodes, &alpha, sub, forwardFilterDim, forwardWeight + (i * forwardFilterDim), 1, &beta, error + (i * uniqueNodes), 1));
	}

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nErrore commesso sui nodi back propagation\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	CHECK(hipFree(sub));
	CHECK(hipFree(padding));

	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);
}

void Convolutional::back_propagation_output(const double * prevOutput, const uint8_t * labels, const int & target, const double & learningRate) {
	// Calcolo dell'errore per ogni nodo
	Kernel::outputErrorK((_alignedNodes / THREADS), THREADS, output, error, labels, target, _nodes);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nErrore commesso sui nodi back propagation output\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);
}

void Convolutional::calcBackPropagation(const double *prevOutput, const double &learningRate) {

	// Applicare derivata della funzione di attivazione
	if (_a == RELU)
		Kernel::derivActReluK(1, _alignedNodes, output, error, _nodes);
	else if (_a == SIGMOID)
		Kernel::derivActSigmoidK(1, _alignedNodes, output, error, _nodes);
	else if (_a == TANH)
		Kernel::derivActTanhK(1, _alignedNodes, output, error, _nodes);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nErrore commesso sui nodi con relativa derivata\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	// Aggiornare i pesi (da mettere in funzione)    
	updateWeights(prevOutput, learningRate);
}

void Convolutional::updateWeights(const double *prevOutput, const double &learningRate) {

	double *sub; // Submatrici

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di filtro
	//(prima genero sottomatrici grandi quanto _filterDim e ne genero tante quante uniqueNodes,
	// ora genero sottomatrici grandi quanto uniqueNodes e ne genero tante quante _filterDim)
	int uniqueNodes = _width * _height;
	const unsigned int subBytes = uniqueNodes * _filterDim * sizeof(double);

	// Alloco submatrice
	CHECK(hipMalloc((void**)&sub, subBytes));

	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	dim3 threadBlocks(_depth, 1, 1);

	// Tanti blocchi quanti sono i nodi dei filtri (_filterWidth * _filterWidth), in questo modo nel kernel sfrutto gli id per righe e colonne delle submatrici
	dim3 numBlocks(_filterWidth, _filterWidth, 1);

	createSubmatrixK(numBlocks, threadBlocks, sub, prevOutput, _prevLayerWidth, _width, _stride, _filterDim);
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore submatrici backpropagation\n\n";
	printFromCudaFormatted(sub, uniqueNodes * _filterDim, _width);
#endif

	// Riempire la matrice temporanea di 0
	CHECK(hipMemset(temp, 0, _wBytes));

	//ora sono in una situazione simile al fully connected
	for (int i = 0; i < _depth; i++) {
		CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, uniqueNodes, _filterDim, &alpha, sub, uniqueNodes, error + (i * uniqueNodes), 1, &beta, temp + (i * _filterDim), 1));
	}

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nMatrice temporanea per aggiornamento pesi\n\n";
	printFromCudaFormatted(temp, _wDim, _filterWidth);
#endif

	// Aggiornamento effettivo dei pesi 
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _wDim, &learningRate, temp, 1, weight, 1));

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nMatrice dei pesi aggiornata\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
#endif

	// Aggiornamento del bias 
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &learningRate, error, 1, bias, 1));

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nVettore del bias aggiornato\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
#endif

	CHECK(hipFree(sub));
}


void Convolutional::deleteCuda() {
	CHECK_CUBLAS(hipblasDestroy(handle));
	CHECK(hipFree(weight));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
	CHECK(hipFree(temp));
}

int Convolutional::_calcOutput(bool withPadding) {
	//PER ORA NON CONSIDERATO CASO IN CUI SI GENERANO ERRORI (padding numero non intero, filtro pi� grande dell'input, stride che non combacia, ecc)
	if (_filterWidth > _prevLayerWidth) {
		std::cerr << "Le dimensioni del filtro superano le dimensioni del livello precedente!!" << std::endl;
		exit(1);
	}

	if (withPadding) {
		_padding = (_filterWidth - 1) / 2;
		return _prevLayerWidth;
	}

	//+(_stride - 1)) serve per aggiornare per eccesso
	return ((_prevLayerWidth - _filterWidth + (_stride - 1)) / _stride) + 1;
}
