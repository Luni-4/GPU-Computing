#include "hip/hip_runtime.h"
#ifdef _WIN32
#include "Windows.h"
#endif

#include <iostream>
#include "Common.h"

// Cuda Kernel
#include "Kernel.h"

#include "Convolutional.h"

__global__ void createSubmatrixBis(double * sub, const double * prevOutput, const int prevLayerWidth, const int filterWidth, const int stride, const int uniqueNodes) {

	// es 24x24 * 1 sottomatrici di 5x5 (ho input di 28x28) 
	// lancio thread di grandezza 5x5 e blocchi di grandezza 24x24
	// tid va da 0 a 24*24*5*5 = 14400
	// blockIdx.x e blockIdx.y rappresentano la sottomatrice
	// blockIdx.z rappresenta la profondit� del livello precedente
	// blockDim.x � il numero di thread nel blocco in orizzontale, 5
	// gridDim.x � il numero di blocchi, 24
	// ad ogni tid corrisponde una posizione dell'input, pTid
	//printf("tid %d, blockId %d, blockIdx.x %d, blockIdx.y %d, gridDim.x %d\n", tid, blockId, blockIdx.x, blockIdx.y, gridDim.x);

	const unsigned int gDim = (gridDim.x * gridDim.y);
	const unsigned int bDim = (blockDim.x * blockDim.y);
	const unsigned int depth = blockIdx.z * gDim * bDim;
	const unsigned int blockId = depth + (blockIdx.y * gridDim.x + blockIdx.x) * bDim;
	const unsigned int tid = blockId + threadIdx.y * blockDim.x + threadIdx.x;

	const unsigned int pDepth = blockIdx.z * prevLayerWidth * prevLayerWidth;
	const unsigned int pBlockId = pDepth + (blockIdx.y * stride) * prevLayerWidth + blockIdx.x * stride;
	const unsigned int pTid = pBlockId + threadIdx.y * prevLayerWidth + threadIdx.x;

	sub[tid] = prevOutput[pTid];
}

void createSubmatrixBisK(dim3 b, dim3 t, double * sub, const double * prevOutput, const int prevLayerWidth, const int filterWidth, const int stride, const int uniqueNodes) {
#ifdef _WIN32
	createSubmatrixBis NvCUDA2(b, t) (sub, prevOutput, prevLayerWidth, filterWidth, stride, uniqueNodes);
#else
	createSubmatrixBis << <b, t >> > (sub, prevOutput, prevLayerWidth, filterWidth, stride, uniqueNodes);
#endif
}

__global__ void createSubmatrix(double * sub, const double * prevOutput, const int prevLayerWidth, const int filterWidth, const int stride, const int uniqueNodes) {

	// es 20x20 * 2 sottomatrici di 5x5 (ho due input di 24x24) 
	// lancio thread di grandezza 2 e blocchi di grandezza 20x20
	// tid va da 0 a 20*20*2 = 800
	// blockIdx.x rappresenta la colonna da cui inizia la submatrice, va da 0 a 20
	// blockIdx.y rappresenta la riga da cui inizia la submatrice, va da 0 a 20
	// blockDim.x � il numero di thread nel blocco, 2
	// gridDim.x � il numero di blocchi, 20
	// printf("tid %d, blockIdx.x %d, blockDim.x %d, blockIdx.y %d, gridDim.x %d\n", tid, blockIdx.x, blockDim.x, blockIdx.y, gridDim.x);

	// Gestione degli indici	
	const unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	const unsigned int tid = blockId + threadIdx.x * uniqueNodes;

	//blockIdx.x rappresenta la colonna da cui inizia la submatrice
	//blockIdx.y rappresenta la riga da cui inizia la submatrice

	//Estraggo submatrici
	if (tid < uniqueNodes * blockDim.x) {
		for (int i = 0; i < filterWidth; i++) {
			memcpy((sub + tid * filterWidth * filterWidth + i * filterWidth), (prevOutput + (threadIdx.x * prevLayerWidth * prevLayerWidth) + (blockIdx.y * stride + i) * prevLayerWidth + blockIdx.x * stride), filterWidth * sizeof(double));
		}
	}
}

void createSubmatrixK(dim3 b, dim3 t, double * sub, const double * prevOutput, const int prevLayerWidth, const int filterWidth, const int stride, const int uniqueNodes) {
#ifdef _WIN32
	createSubmatrix NvCUDA2(b, t) (sub, prevOutput, prevLayerWidth, filterWidth, stride, uniqueNodes);
#else
	createSubmatrix << <b, t >> > (sub, prevOutput, prevLayerWidth, filterWidth, stride, uniqueNodes);
#endif
}

__global__ void zeroPaddingBis(double * error, const double * forwardError, const int forwardErrorWidth, const int forwardFilterWidth) {
	//threadIdx.y rappresenta la riga 
	//threadIdx.x rappresenta la colonna
	const unsigned int paddingLeft = forwardFilterWidth - 1;
	const unsigned int widthWithPadding = forwardErrorWidth + (paddingLeft * 2);
	const unsigned int paddingTop = blockIdx.z * widthWithPadding * widthWithPadding + paddingLeft * widthWithPadding;

	const unsigned int tid = paddingTop + threadIdx.y * widthWithPadding + paddingLeft + threadIdx.x;
	const unsigned int pTid = blockIdx.z * forwardErrorWidth * forwardErrorWidth + threadIdx.y * forwardErrorWidth + threadIdx.x;

	error[tid] = forwardError[pTid];
}

void zeroPaddingBisK(dim3 b, dim3 t, double * error, const double * forwardError, const int forwardErrorWidth, const int forwardFilterWidth) {
#ifdef _WIN32
	zeroPaddingBis NvCUDA2(b, t) (error, forwardError, forwardErrorWidth, forwardFilterWidth);
#else
	zeroPaddingBis << <b, t >> > (error, forwardError, forwardErrorWidth, forwardFilterWidth);
#endif
}

__global__ void zeroPadding(double * error, const double * forwardError, const int forwardErrorWidth, const int forwardFilterWidth) {
	//blockIdx.y rappresenta la riga 
	const unsigned int paddingLeft = forwardFilterWidth - 1;
	const unsigned int widthWithPadding = forwardErrorWidth + (paddingLeft * 2);
	const unsigned int tid = ((blockIdx.y + paddingLeft) * widthWithPadding) + (widthWithPadding * widthWithPadding * threadIdx.x) + paddingLeft;

	memcpy((error + tid), (forwardError + blockIdx.y * forwardErrorWidth + forwardErrorWidth * forwardErrorWidth * threadIdx.x), (forwardErrorWidth * sizeof(double)));
}

void zeroPaddingK(dim3 b, dim3 t, double * error, const double * forwardError, const int forwardErrorWidth, const int forwardFilterWidth) {
#ifdef _WIN32
	zeroPadding NvCUDA2(b, t) (error, forwardError, forwardErrorWidth, forwardFilterWidth);
#else
	zeroPadding << <b, t >> > (error, forwardError, forwardErrorWidth, forwardFilterWidth);
#endif
}

__global__ void rot180Bis(const double * forwardWeight, double * forwardWeightRot, int filterDim) {

	// es 2 filtri di 5x5
	// per ora lancio thread di grandezza 2 e blocchi di grandezza 5x5
	// tid va da 0 a 5*5*2 = 50
	// blockIdx.x rappresenta la colonna da cui inizia la submatrice, va da 0 a 5
	// blockIdx.y rappresenta la riga da cui inizia la submatrice, va da 0 a 5
	// blockDim.x � il numero di thread nel blocco, 2
	// gridDim.x � il numero di blocchi, 5
	//printf("tid %d, threadIdx.x %d, threadIdx.x %d, blockIdx.x %d, blockDim.x %d, blockIdx.y %d, blockDim.y %d, gridDim.x %d\n", tid, threadIdx.x, threadIdx.y, blockIdx.x, blockDim.x, blockIdx.y, blockDim.y, gridDim.x);

	// Gestione degli indici
	const unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	const unsigned int tid = blockId + (threadIdx.x + threadIdx.y * blockDim.x) * filterDim;

	const int plus = filterDim + (threadIdx.x + threadIdx.y *  blockDim.x) * filterDim - 1;
	forwardWeightRot[tid] = forwardWeight[plus - blockId];
}

void rot180BisK(dim3 b, dim3 t, const double * forwardWeight, double * forwardWeightRot, int filterDim) {
#ifdef _WIN32
	rot180Bis NvCUDA2(b, t) (forwardWeight, forwardWeightRot, filterDim);
#else
	rot180Bis << <b, t >> > (forwardWeight, forwardWeightRot, filterDim);
#endif
}

__global__ void rot180(const double * forwardWeight, double * forwardWeightRot, int filterDim) {

	// es 2 filtri di 5x5
	// per ora lancio thread di grandezza 2 e blocchi di grandezza 5x5
	// tid va da 0 a 5*5*2 = 50
	// blockIdx.x rappresenta la colonna da cui inizia la submatrice, va da 0 a 5
	// blockIdx.y rappresenta la riga da cui inizia la submatrice, va da 0 a 5
	// blockDim.x � il numero di thread nel blocco, 2
	// gridDim.x � il numero di blocchi, 5
	//printf("tid %d, threadIdx.x %d, threadIdx.x %d, blockIdx.x %d, blockDim.x %d, blockIdx.y %d, blockDim.y %d, gridDim.x %d\n", tid, threadIdx.x, threadIdx.y, blockIdx.x, blockDim.x, blockIdx.y, blockDim.y, gridDim.x);

	// Gestione degli indici
	const unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	const unsigned int tid = blockId + (threadIdx.x + threadIdx.y * blockDim.x) * filterDim;


	const int plus = filterDim + (threadIdx.x + threadIdx.y *  blockDim.x) * filterDim - 1;
	memcpy((forwardWeightRot + tid), (forwardWeight + plus - blockId), (sizeof(double)));
}

void rot180K(dim3 b, dim3 t, const double * forwardWeight, double * forwardWeightRot, int filterDim) {
#ifdef _WIN32
	rot180 NvCUDA2(b, t) (forwardWeight, forwardWeightRot, filterDim);
#else
	rot180 << <b, t >> > (forwardWeight, forwardWeightRot, filterDim);
#endif
}

Convolutional::Convolutional(const int &filterWidth, const int &depth, const int &stride, const ActFctType &a)
	: LayerDefinition(0, 0, depth, CONVOLUTIONAL, a) {
	this->_filterWidth = filterWidth;
	this->_filterDim = filterWidth * filterWidth;
	this->_depth = depth;
	this->_stride = stride;
	this->_padding = 0;
}

Convolutional::~Convolutional() {
}

std::vector<double> Convolutional::getWeights(void) {
	std::vector<double> wCPU(_wDim);
	CHECK(hipMemcpy(&wCPU[0], weight, _wBytes, hipMemcpyDeviceToHost));
	return wCPU;
}

std::vector<double> Convolutional::getBias(void) {
	std::vector<double> bCPU(_nodes);
	CHECK(hipMemcpy(&bCPU[0], bias, _nodes * sizeof(double), hipMemcpyDeviceToHost));
	return bCPU;
}

int Convolutional::getPredictionIndex(void) {
	int maxIndex;

	// Individuare indice (classe) che corrisponde al valore massimo di output
	CHECK_CUBLAS(
		hipblasIdamax(handle, _nodes, output, 1, &maxIndex));

	return maxIndex - 1;
}

void Convolutional::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {
	_prevLayerWidth = prevLayerWidth;
	_prevLayerDepth = prevLayerDepth;

	//numero di nodi dipende da filtro e nodi livello precedente
	//width
	_width = _calcOutput(false);
	//height
	_height = _calcOutput(false);
	//depth = numero di filtri

	this->_nodes = _width * _height * _depth;
	_alignedNodes = ALIGN_UP(_nodes, THREADS);

	_uniqueNodes = _width * _height;

#ifdef DEBUG
	std::cout << "dimensioni output del livello: " << _width << " - " << _height << " - " << _depth << std::endl;
#endif

	//Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));

	// Impostazioni della cache
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	// Dimensione matrice dei pesi
	_wDim = _filterDim * prevLayerDepth * _depth;

	// Dimensione matrice dei pesi in byte
	_wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);

#ifdef DEBUG
	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);
#endif

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, _wBytes));
	CHECK(hipMalloc((void**)&weightRot, _wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	CHECK(hipMalloc((void**)&error, Bytes));
	CHECK(hipMalloc((void**)&errorRot, Bytes));
	CHECK(hipMalloc((void**)&tempWeight, _wBytes));
	CHECK(hipMalloc((void**)&tempOutput, Bytes));

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(_filterDim, THREADS);

	// Tanti blocchi quanto sono i filtri e la profondit� del layer precedente
	dim3 numBlocks(_depth, prevLayerDepth, 1);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi che compongono i filtri
	dim3 threadBlocks(aligned, 1, 1);

	// Inizializza array per numeri casuali
	hiprandStateXORWOW_t *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _nodes * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandStateXORWOW_t)));

	// Inizializzare i weight del livello
	Kernel::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// Inizializzare i bias del livello
	Kernel::initBiasK((_alignedNodes / THREADS), THREADS, bias, _nodes, devStates);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore dei pesi\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
	std::cout << "\n\nValore dei bias\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
	std::cout << "\n\n\n\n";
#endif

	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	threadBlocks = dim3(_depth, _prevLayerDepth, 1);

	// Tanti blocchi quante sono le righe e le colonne di forwardError
	numBlocks = dim3(_filterWidth, _filterWidth, 1);

#ifdef BIS
	rot180BisK(numBlocks, threadBlocks, weight, weightRot, _filterDim);
#else
	rot180K(numBlocks, threadBlocks, weight, weightRot, _filterDim);
#endif

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore dei pesi ruotati\n\n";
	printFromCudaFormatted(weightRot, _wDim, _filterWidth);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}

void Convolutional::forward_propagation(const double * prevOutput) {

#ifdef DEBUG
	std::cout << "\n\nValore dell'input\n\n";
	printFromCudaFormatted(prevOutput, _prevLayerWidth * _prevLayerWidth * _prevLayerDepth, _prevLayerWidth);
#endif

	double *sub; // Submatrici

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo che compone un blocco di output * la profondit� del livello precedente e grande quanto un filtro
	const unsigned int subBytes = _uniqueNodes * _prevLayerDepth * _filterDim * sizeof(double);

	// Alloco submatrice
	CHECK(hipMalloc((void**)&sub, subBytes));

#ifdef BIS
	// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
	dim3 threadBlocks(_filterWidth, _filterWidth, 1);

	// Tanti blocchi quanti sono i nodi in output e il depth del livello precedente
	dim3 numBlocks(_width, _height, _prevLayerDepth);

	createSubmatrixBisK(numBlocks, threadBlocks, sub, prevOutput, _prevLayerWidth, _filterWidth, _stride, _uniqueNodes);
#else
	// Blocchi bidimensionali contenenti tanti thread quanti il depth del livello precedente
	dim3 threadBlocks(_prevLayerDepth, 1, 1);

	// Tanti blocchi quanti sono i nodi in output (width * height), in questo modo nel kernel sfrutto gli id per righe e colonne delle submatrici
	dim3 numBlocks(_width, _height, 1);

	createSubmatrixK(numBlocks, threadBlocks, sub, prevOutput, _prevLayerWidth, _filterWidth, _stride, _uniqueNodes);
#endif

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici\n\n";
	printFromCudaFormatted(sub, _uniqueNodes * _prevLayerDepth * _filterDim, _filterWidth);
#endif

	//ora sono in una situazione simile al fully connected
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {
			CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _filterDim, _uniqueNodes, &alpha, sub + (j * _uniqueNodes), _filterDim, weightRot + (i * _filterDim * _prevLayerDepth) + (j * _filterDim), 1, &beta, output + (i * _uniqueNodes), 1));
		}
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore output senza bias\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Somma con il bias
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &alpha, bias, 1, output, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _nodes, &alpha, bias, 1, &alpha, output, 1, output, 1));

#ifdef DEBUG
	std::cout << "\n\nValore output prima di funzione di attivazione\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Applicare funzione di attivazione
	if (_a == RELU)
		Kernel::actReluK((_alignedNodes / THREADS), THREADS, output, tempOutput, _nodes);
	else if (_a == SIGMOID)
		Kernel::actSigmoidK((_alignedNodes / THREADS), THREADS, output, _nodes);
	else if (_a == TANH)
		Kernel::actTanhK((_alignedNodes / THREADS), THREADS, output, _nodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore output *************************************************\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	CHECK(hipFree(sub));
}

void Convolutional::calcError(double *prevError, const int &prevNodes) {

	//prev error � l'errore del livello precedente che devo riempire, 
	//error � l'errore che ho usato al passo precedente (non ruotato) quando sono passato da questo livello

#ifdef DEBUG
	std::cout << "\n\error in calc error\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	// matrice temporanea inizializzata a 0 per zero padding
	double *padding;
	const int paddingWidth = (_filterWidth - 1) * 2 + _width;
	const int uniquePadding = paddingWidth * paddingWidth;
	const int paddingSize = uniquePadding * _depth; //come output 
	CHECK(hipMalloc((void**)&padding, paddingSize * sizeof(double)));
	CHECK(hipMemset(padding, 0, paddingSize * sizeof(double)));

#ifdef BIS
	// Blocchi bidimensionali contenenti tanti thread quanti sono i nodi in output
	dim3 threadBlocks(_height, _width, 1);

	// Tanti blocchi quanto il numero di filtri
	dim3 numBlocks(1, 1, _depth);

	zeroPaddingBisK(numBlocks, threadBlocks, padding, error, _width, _filterWidth);
#else
	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	dim3 threadBlocks(_depth, 1, 1);

	// Tanti blocchi quante sono le righe di forwardError, in questo modo nel kernel sfrutto gli id.y per righe
	dim3 numBlocks(1, _width, 1);

	zeroPaddingK(numBlocks, threadBlocks, padding, error, _width, _filterWidth);
#endif

#ifdef DEBUG
	std::cout << "\n\nerror con zero padding\n\n";
	printFromCudaFormatted(padding, paddingSize, paddingWidth);
#endif

	double *sub; // Submatrici

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di output di L-1
	const int prevUniqueNodes = prevNodes / _prevLayerDepth;
	const unsigned int subBytes = prevUniqueNodes * _depth * _filterDim * sizeof(double);

	// Alloco submatrice
	CHECK(hipMalloc((void**)&sub, subBytes));

#ifdef BIS
	// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
	threadBlocks = dim3(_filterWidth, _filterWidth, 1);

	// Tanti blocchi quanti sono i nodi in input e il depth del livello precedente
	numBlocks = dim3(sqrt(prevUniqueNodes), sqrt(prevUniqueNodes), _prevLayerDepth);

	createSubmatrixBisK(numBlocks, threadBlocks, sub, padding, paddingWidth, _filterWidth, _stride, prevUniqueNodes);
#else
	// Blocchi bidimensionali contenenti tanti thread quanti il depth del livello precedente
	threadBlocks = dim3(_depth, 1, 1);

	// Tanti blocchi quanti sono i nodi in output (width * height), in questo modo nel kernel sfrutto gli id per righe e colonne delle submatrici
	numBlocks = dim3(sqrt(prevUniqueNodes), sqrt(prevUniqueNodes), 1);

	createSubmatrixK(numBlocks, threadBlocks, sub, padding, paddingWidth, _filterWidth, _stride, prevUniqueNodes);
#endif

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici zero padding\n\n";
	printFromCudaFormatted(sub, prevUniqueNodes * _depth * _filterDim, _filterWidth);
#endif

	//ora sono in una situazione simile alla convoluzione
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {
			CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _filterDim, prevUniqueNodes, &alpha, sub + (i * prevUniqueNodes), _filterDim, weightRot + ((i + j * _depth) * _filterDim), 1, &beta, prevError + (j * prevUniqueNodes), 1));
		}
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi back propagation\n\n";
	printFromCudaFormatted(prevError, prevNodes, sqrt(prevUniqueNodes));
#endif

	CHECK(hipFree(sub));
	CHECK(hipFree(padding));
}

void Convolutional::back_propagation(const double *prevOutput, const double &learningRate) {
	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);
}

void Convolutional::back_propagation_output(const double * prevOutput, const uint8_t * labels, const int & target, const double & learningRate) {
	// Calcolo dell'errore per ogni nodo
	Kernel::outputErrorK((_alignedNodes / THREADS), THREADS, output, error, labels, target, _nodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi back propagation output\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);
}

void Convolutional::calcBackPropagation(const double *prevOutput, const double &learningRate) {

	// Blocchi bidimensionali contenenti tanti thread quanti il numero di depth in output
	dim3 threadBlocks(_depth, 1, 1);

	// Tanti blocchi quante sono le righe e le colonne di error
	dim3 numBlocks(_width, _height, 1);

#ifdef BIS
	rot180BisK(numBlocks, threadBlocks, error, errorRot, _uniqueNodes);
#else
	rot180K(numBlocks, threadBlocks, error, errorRot, _uniqueNodes);
#endif

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore error ruotato\n\n";
	printFromCudaFormatted(errorRot, _nodes, _width);
#endif

	// Applicare derivata della funzione di attivazione
	if (_a == RELU)
		Kernel::derivActReluK((_alignedNodes / THREADS), THREADS, errorRot, tempOutput, _nodes);
	else if (_a == SIGMOID)
		Kernel::derivActSigmoidK((_alignedNodes / THREADS), THREADS, output, errorRot, _nodes);
	else if (_a == TANH)
		Kernel::derivActTanhK((_alignedNodes / THREADS), THREADS, output, errorRot, _nodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi con relativa derivata\n\n";
	printFromCudaFormatted(errorRot, _nodes, _width);
#endif

	updateWeights(prevOutput, learningRate);
}

void Convolutional::updateWeights(const double *prevOutput, const double &learningRate) {

	double *sub; // Submatrici

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di filtro
	//(prima genero sottomatrici grandi quanto _filterDim e ne genero tante quante uniqueNodes,
	// ora genero sottomatrici grandi quanto uniqueNodes e ne genero tante quante _filterDim)
	const unsigned int subBytes = _uniqueNodes * _prevLayerDepth * _filterDim * sizeof(double);

	// Alloco submatrice
	CHECK(hipMalloc((void**)&sub, subBytes));

#ifdef BIS
	// Blocchi tridimensionali contenenti tanti thread quanti sono i nodi in output
	dim3 threadBlocks(_width, _height, 1);

	// Tanti blocchi quanti la grandezza dei filtri e il depth del livello precedente
	dim3 numBlocks(_filterWidth, _filterWidth, _prevLayerDepth);

	createSubmatrixBisK(numBlocks, threadBlocks, sub, prevOutput, _prevLayerWidth, _width, _stride, _filterDim);
#else
	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	dim3 threadBlocks(_prevLayerDepth, 1, 1);

	// Tanti blocchi quanti sono i nodi dei filtri (_filterWidth * _filterWidth), in questo modo nel kernel sfrutto gli id per righe e colonne delle submatrici
	dim3 numBlocks(_filterWidth, _filterWidth, 1);

	// come in forward ma sostituendo filterWidth con width e uniquenodes con filterdim
	createSubmatrixK(numBlocks, threadBlocks, sub, prevOutput, _prevLayerWidth, _width, _stride, _filterDim);
#endif

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici backpropagation\n\n";
	printFromCudaFormatted(sub, _uniqueNodes * _filterDim, _width);
#endif

	// Riempire la matrice temporanea di 0
	CHECK(hipMemset(tempWeight, 0, _wBytes));

	//ora sono in una situazione simile al fully connected
	double backAlpha = 1.0 / _uniqueNodes;
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {
			CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _uniqueNodes, _filterDim, &backAlpha, sub + (j * _filterDim), _uniqueNodes, errorRot + (i * _uniqueNodes), 1, &beta, tempWeight + ((i + j * _depth) * _filterDim), 1));
		}
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice temporanea per aggiornamento pesi\n\n";
	printFromCudaFormatted(tempWeight, _wDim, _filterWidth);
#endif

	// Aggiornamento effettivo dei pesi 
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _wDim, &learningRate, tempWeight, 1, weight, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, _wDim, _depth, &learningRate, tempWeight, _wDim, &alpha, weight, _wDim, weight, _wDim));

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice dei pesi aggiornata\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
#endif

	// Ruoto subito i pesi aggiornati per poi usarli nella backpropagation al livello L-1
	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	threadBlocks = dim3(_depth, _prevLayerDepth, 1);

	// Tanti blocchi quante sono le righe e le colonne di forwardError
	numBlocks = dim3(_filterWidth, _filterWidth, 1);

#ifdef BIS
	rot180BisK(numBlocks, threadBlocks, weight, weightRot, _filterDim);
#else
	rot180K(numBlocks, threadBlocks, weight, weightRot, _filterDim);
#endif

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore dei pesi ruotati\n\n";
	printFromCudaFormatted(weightRot, _wDim, _filterWidth);
#endif

	// Aggiornamento del bias 
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &learningRate, errorRot, 1, bias, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _nodes, &learningRate, errorRot, 1, &alpha, bias, 1, bias, 1));

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nVettore del bias aggiornato\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
#endif

	CHECK(hipFree(sub));
}

void Convolutional::deleteCuda() {
	CHECK_CUBLAS(hipblasDestroy(handle));
	CHECK(hipFree(weight));
	CHECK(hipFree(weightRot));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
	CHECK(hipFree(errorRot));
	CHECK(hipFree(tempWeight));
	CHECK(hipFree(tempOutput));
}

void Convolutional::printW() {
	printFromCudaFormatted(weight, _wDim, _filterWidth);
	//printFromCudaFormatted(bias, _nodes, _width);
}

int Convolutional::_calcOutput(bool withPadding) {
	//PER ORA NON CONSIDERATO CASO IN CUI SI GENERANO ERRORI (padding numero non intero, filtro pi� grande dell'input, stride che non combacia, ecc)
	if (_filterWidth > _prevLayerWidth) {
		std::cerr << "Le dimensioni del filtro superano le dimensioni del livello precedente!!" << std::endl;
		exit(1);
	}

	if (withPadding) {
		_padding = (_filterWidth - 1) / 2;
		return _prevLayerWidth;
	}

	//+(_stride - 1)) serve per aggiornare per eccesso
	return ((_prevLayerWidth - _filterWidth + (_stride - 1)) / _stride) + 1;
}
