#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>

// Cuda Kernel
#include "Kernel.h"

#include "Common.h"
#include "Convolutional.h"

#ifdef _WIN32
#include "Windows.h"
#endif

__global__ void createSubmatrix(double * sub, const double * prev, const int prevLayerWidth, const int filterWidth, const int nodes) {

	// Gestione degli indici	
	const unsigned int blockId = blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
	const unsigned int tid = blockId + threadIdx.x;

	//tid rappresenta il nodo di output
	//blockIdx.x rappresenta la colonna da cui inizia la submatrice
	//blockIdx.y rappresenta la riga da cui inizia la submatrice

	//Estraggo submatrici
	if (tid < nodes) {
		for (int i = 0; i < filterWidth; i++) {
			memcpy((sub + i * filterWidth + tid * filterWidth * filterWidth), (prev + blockIdx.x + (blockIdx.y + i) * prevLayerWidth), filterWidth * sizeof(double));
		}
	}
}

void createSubmatrixK(dim3 t, dim3 b, double * sub, const double * prev, const int prevLayerWidth, const int filterWidth, const int nodes) {
#ifdef _WIN32
	createSubmatrix NvCUDA2(t, b) (sub, prev, prevLayerWidth, filterWidth, nodes);
#else
	createSubmatrix << <t, b >> > (sub, prev, prevLayerWidth, filterWidth, nodes);
#endif
}

Convolutional::Convolutional(const int &filterWidth, const int &depth, const int &stride, const ActFctType &a)
	: LayerDefinition(0, 0, depth, CONVOLUTIONAL, a) {
	this->_depth = depth;
	this->_filterWidth = filterWidth;
	this->_stride = stride;
	this->_padding = 0;
}

Convolutional::~Convolutional() {
}

int Convolutional::getLayerNodeCount() {
	return 0;
}

int Convolutional::getWeightCount(const int & prevLayerNode) {
	return 0;
}

std::vector<double> Convolutional::getWeights() {
	return std::vector<double>();
}

std::vector<double> Convolutional::getBias() {
	return std::vector<double>();
}

void Convolutional::forward_propagation(const double * prev) {
#ifdef DEBUG
	std::cout << "\n\nValore dell'input\n\n";
	printFromCudaFormatted(prev, _prevLayerWidth * _prevLayerWidth, _prevLayerWidth);
#endif

	double *sub; // Submatrici

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di output
	const unsigned int subBytes = _nodes * _filterWidth * _filterWidth * sizeof(double);

	// Alloco submatrice
	CHECK(hipMalloc((void**)&sub, subBytes));

	// Blocchi bidimensionali contenenti tanti thread quanti i numeri di filtri
	dim3 threadBlocks(_depth, 1, 1);

	// Tanti blocchi quanti sono i nodi in output (width * height), in questo modo nel kernel sfrutto gli id per righe e colonne delle submatrici
	dim3 numBlocks(_width, _height, 1);

	createSubmatrixK(numBlocks, threadBlocks, sub, prev, _prevLayerWidth, _filterWidth, _nodes);
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore submatrici\n\n";
	printFromCudaFormatted(sub, _nodes * _filterWidth * _filterWidth, _filterWidth);
	//printFromCudaFormatted(sub, _filterWidth * _filterWidth, _filterWidth);
#endif

	//Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));

	//Fattori dei prodotti
	const double alpha = 1.0f;
	const double beta = 0.0f;

	//ora sono in una situazione simile al fully connected
	CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _filterWidth * _filterWidth, _nodes, &alpha, sub, _filterWidth * _filterWidth, weight, 1, &beta, output, 1));

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore output senza bias\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Somma con il bias
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &alpha, bias, 1, output, 1));

#ifdef DEBUG
	std::cout << "\n\nValore output prima di funzione di attivazione\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Applicare funzione di attivazione
	if (_a == RELU)
		Kernel::actReluK(1, _alignedNodes, output, _nodes);
	else if (_a == SIGMOID)
		Kernel::actSigmoidK(1, _alignedNodes, output, _nodes);
	else if (_a == TANH)
		Kernel::actTanhK(1, _alignedNodes, output, _nodes);

#ifdef DEBUG
	std::cout << "\n\nValore output\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

	CHECK(hipFree(sub));
}

void Convolutional::back_propagation() {
}

void Convolutional::back_propagation_output(const double * prev, const uint8_t * labels, const int & target, const double & learningRate) {
}

void Convolutional::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {
	_prevLayerWidth = prevLayerWidth;

	//numero di nodi dipende da filtro e nodi livello precedente
	//width
	_width = _calcOutput(prevLayerWidth, false);
	//height
	_height = _calcOutput(prevLayerHeight, false);
	//depth = numero di filtri

	this->_nodes = _width * _height * _depth;

#ifdef DEBUG
	std::cout << "dimensioni output del livello: " << _width << " - " << _height << " - " << _depth << std::endl;
#endif

	// Dimensione matrice dei pesi
	_wDim = _filterWidth * _filterWidth * prevLayerDepth * _depth;

	// Dimensione matrice dei pesi in byte
	const unsigned int wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);

	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	CHECK(hipMalloc((void**)&error, Bytes));

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(_filterWidth * _filterWidth);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi che compongono i filtri
	dim3 threadBlocks(aligned, 1, 1);

	// Tanti blocchi quanto sono i filtri e la profondit� del layer precedente
	dim3 numBlocks(_depth, prevLayerDepth, 1);

	// Inizializza array per numeri casuali
	hiprandState *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _depth * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandState)));

	// Inizializzare i weight del livello
	Kernel::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

	// Convertire il numero di filtri in un multiplo di 32
	//datascience.stackexchange.com/questions/11853/question-about-bias-in-convolutional-networks
	_alignedNodes = ALIGN_UP(_nodes);

	// Inizializzare i bias del livello
	Kernel::initBiasK(1, _alignedNodes, bias, _nodes, devStates);

	// CPU deve attendere che esecuzione della funzione finisca
	CHECK(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "\n\nValore dei pesi\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
	std::cout << "\n\nValore dei bias\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}

void Convolutional::deleteCuda() {
	CHECK_CUBLAS(hipblasDestroy(handle));
	CHECK(hipFree(weight));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
}

int Convolutional::_calcOutput(int prevLayerWidth, bool withPadding) {
	//PER ORA NON CONSIDERATO CASO IN CUI SI GENERANO ERRORI (padding numero non intero, filtro pi� grande dell'input, stride che non combacia, ecc)
	if (withPadding) {
		_padding = (_filterWidth - 1) / 2;
		return prevLayerWidth;
	}
	return ((prevLayerWidth - _filterWidth + 0) / _stride) + 1;
}
