#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <memory>

// Librerie Cuda

#ifdef _WIN32
#include "Windows.h"
#endif

#ifdef DEBUG
#include "Common.h"
#endif

// Librerie di progetto
#include "Mnist.h"
#include "FullyConnected.h"
#include "Convolutional.h"
#include "Network.h"

#ifdef DEBUG

void test_input() {
	// Leggere i dati
	Data* d = new Mnist("../data/");

	// Lettura dati di training
	d->readTrainData();

	delete d;
}


void test_fully() {
	LayerDefinition* layer = new FullyConnected(10, RELU);

	// Stampa dei vari parametri del livello
	printf("Larghezza layer: %d\n", layer->getWidth());
	printf("Altezza layer: %d\n", layer->getHeight());
	printf("Tipo di layer: %d\n", layer->getLayerType());
	printf("Funzione di attivazione: %d", layer->getActivationFunction());

	// Definizione di un'immagine di 4 pixel rgb
	layer->defineCuda(2, 2, 3);

	double *inp;

	CHECK(hipMalloc((void**)&inp, 12 * sizeof(double)));

	CHECK(hipMemset(inp, 0, 12 * sizeof(double)));

	std::cout << "\n\n\nImmagine di input RGB\n\n";
	printFromCuda(inp, 12);

	// Passare l'immagine ed eseguire prodotto più aggiunta del bias
	layer->forward_propagation(inp);

	std::vector<double> w = layer->getWeights();

	std::cout << "\n\n\nNumero dei pesi: ";
	std::cout << w.size();
	std::cout << "\n\nPesi\n\n";

	for (auto t : w)
		std::cout << t << std::endl;

	layer->deleteCuda();

	CHECK(hipFree(inp));

	delete layer;
}

#endif

int main() {

#ifdef DEBUG
	//test_input();

	//test_fully();
#endif

	// Leggere i dati
	std::unique_ptr<Data> d(new Mnist("../data/"));

	// Vettore contenente i livelli della rete
	std::vector<std::unique_ptr<LayerDefinition>> layers;

	// Inizializzare i livelli	
#ifdef _WIN32	
	layers.emplace_back(new FullyConnected(10, RELU));
	//layers.emplace_back(new Convolutional(5, 1, 1, RELU));
#else
	layers.emplace_back(new FullyConnected(10, RELU));
#endif

	// Creare la rete
	Network nn(layers);

	// Training
	nn.train(d.get(), 20, 0.5);

	// Test
	//nn.predict(//param);

#ifdef _WIN32
	system("pause");
#endif

}
