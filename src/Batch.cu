#include "hip/hip_runtime.h"
#ifdef _WIN32
#include "Windows.h"
#endif

#include <iostream>
#include "Common.h"

// Cuda Kernel
#include "KernelCPU.h"

#include "Batch.h"

Batch::Batch(const int &filterWidth, const int &depth, const int &stride)
	: LayerDefinition(0, 0, depth, BATCH, NONE) {
	this->_filterWidth = filterWidth;
	this->_filterDim = filterWidth * filterWidth;
	this->_depth = depth;
	this->_stride = stride;
	this->_padding = 0;
}

Batch::~Batch() {
}

std::vector<double> Batch::getWeights(void) {
	std::vector<double> wCPU(_wDim);
	CHECK(hipMemcpy(&wCPU[0], weight, _wBytes, hipMemcpyDeviceToHost));
	return wCPU;
}

std::vector<double> Batch::getBias(void) {
	std::vector<double> bCPU(_nodes);
	CHECK(hipMemcpy(&bCPU[0], bias, _nodes * sizeof(double), hipMemcpyDeviceToHost));
	return bCPU;
}

int Batch::getPredictionIndex(void) {
	int maxIndex;

	// Individuare indice (classe) che corrisponde al valore massimo di output
	CHECK_CUBLAS(
		hipblasIdamax(handle, _nodes, output, 1, &maxIndex));

	return maxIndex - 1;
}

void Batch::defineCuda(const int &prevLayerWidth, const int &prevLayerHeight, const int &prevLayerDepth) {
	_prevLayerWidth = prevLayerWidth;
	_prevLayerDepth = prevLayerDepth;

	//numero di nodi dipende da filtro e nodi livello precedente
	//width
	_width = _calcOutput(false);
	//height
	_height = _calcOutput(false);
	//depth = numero di filtri

	this->_nodes = _width * _height * _depth;
	_alignedNodes = ALIGN_UP(_nodes, THREADS);

	_uniqueNodes = _width * _height;

#ifdef DEBUG
	std::cout << "dimensioni output del livello: " << _width << " - " << _height << " - " << _depth << std::endl;
#endif

	//Creare l'handle di cuBLAS
	CHECK_CUBLAS(hipblasCreate(&handle));

	// Impostazioni della cache
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	// Dimensione matrice dei pesi
	_wDim = _filterDim * prevLayerDepth * _depth;

	// Dimensione matrice dei pesi in byte
	_wBytes = _wDim * sizeof(double);

	// Dimensione bias, output, error
	const unsigned int Bytes = _nodes * sizeof(double);

#ifdef DEBUG
	// Impostazione buffer che gestisce il printf in Cuda
	size_t sz = 1048576 * 1000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);
#endif

	// Allocare le matrici
	CHECK(hipMalloc((void**)&weight, _wBytes));
	CHECK(hipMalloc((void**)&weightRot, _wBytes));
	CHECK(hipMalloc((void**)&bias, Bytes));
	CHECK(hipMalloc((void**)&output, Bytes));
	CHECK(hipMalloc((void**)&error, Bytes));
	CHECK(hipMalloc((void**)&tempWeight, _wBytes));
	CHECK(hipMalloc((void**)&tempOutput, Bytes));

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo che compone un blocco di output * la profondit� del livello precedente e grande quanto un filtro
	unsigned int subBytes = _uniqueNodes * _prevLayerDepth * _filterDim * sizeof(double);
	CHECK(hipMalloc((void**)&subForward, subBytes));

	const int prevUniqueNodes = _prevLayerWidth * _prevLayerWidth;
	subBytes = prevUniqueNodes * _depth * _filterDim * sizeof(double);
	CHECK(hipMalloc((void**)&subCalcError, subBytes));

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di filtro
	//(prima genero sottomatrici grandi quanto _filterDim e ne genero tante quante uniqueNodes,
	// ora genero sottomatrici grandi quanto uniqueNodes e ne genero tante quante _filterDim)
	subBytes = _uniqueNodes * _prevLayerDepth * _filterDim * sizeof(double);
	CHECK(hipMalloc((void**)&subBack, subBytes));

	// matrice temporanea inizializzata a 0 per zero padding
	paddingWidth = (_filterWidth - 1) * 2 + _width;
	const int uniquePadding = paddingWidth * paddingWidth;
	paddingSize = uniquePadding * _depth; //come output 
	CHECK(hipMalloc((void**)&padding, paddingSize * sizeof(double)));
	CHECK(hipMemset(padding, 0, paddingSize * sizeof(double)));

#ifdef DEBUG
	std::cout << "Memoria allocata \n" << std::endl;
#endif

	// Rendere i blocchi multipli di 32
	const int aligned = ALIGN_UP(_filterDim, THREADS);

	// Tanti blocchi quanto sono i filtri e la profondit� del layer precedente
	dim3 numBlocks(_depth, prevLayerDepth, 1);

	// Blocchi bidimensionali contenenti tanti thread quanti i nodi che compongono i filtri
	dim3 threadBlocks(aligned, 1, 1);

	// Inizializza array per numeri casuali
	hiprandStateXORWOW_t *devStates;

	// Numero di sequenze diverse per il rand
	const int numRand = _nodes * prevLayerDepth * aligned;

	// Alloca la memoria
	CHECK(hipMalloc((void **)&devStates, numRand * sizeof(hiprandStateXORWOW_t)));

	// Inizializzare i weight del livello
	Kernel::initWeightK(numBlocks, threadBlocks, weight, _wDim, devStates);

	// Inizializzare i bias del livello
	Kernel::initBiasK((_alignedNodes / THREADS), THREADS, bias, _nodes, devStates);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore dei pesi\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
	std::cout << "\n\nValore dei bias\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
	std::cout << "\n\n\n\n";
#endif

	// Distrugge gli stati
	CHECK(hipFree(devStates));
}

void Batch::forward_propagation(const double * prevOutput) {

#ifdef DEBUG
	std::cout << "\n\nValore dell'input\n\n";
	printFromCudaFormatted(prevOutput, _prevLayerWidth * _prevLayerWidth * _prevLayerDepth, _prevLayerWidth);
#endif

	// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
	dim3 threadBlocks(_filterWidth, _filterWidth, 1);

	// Tanti blocchi quanti sono i nodi in output e il depth del livello precedente
	dim3 numBlocks(_width, _height, _prevLayerDepth);

	Kernel::createSubmatrixBisK(numBlocks, threadBlocks, subForward, prevOutput, _prevLayerWidth, _filterWidth, _stride, _uniqueNodes);

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici\n\n";
	printFromCudaFormatted(subForward, _uniqueNodes * _prevLayerDepth * _filterDim, _filterWidth);
#endif

	//ora sono in una situazione simile al fully connected
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {
			CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _filterDim, _uniqueNodes, &alpha, subForward + (j * _uniqueNodes), _filterDim, weight + (i * _filterDim * _prevLayerDepth) + (j * _filterDim), 1, &beta, output + (i * _uniqueNodes), 1));
		}
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore output senza bias\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif

	// Somma con il bias
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &alpha, bias, 1, output, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _nodes, &alpha, bias, 1, &alpha, output, 1, output, 1));


#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore output *************************************************\n\n";
	printFromCudaFormatted(output, _nodes, _width);
#endif
}

void Batch::calcError(double *prevError, const int &prevNodes) {

	//prev error � l'errore del livello precedente che devo riempire, 
	//error � l'errore che ho usato al passo precedente (non ruotato) quando sono passato da questo livello

#ifdef DEBUG
	std::cout << "\n\error in calc error\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	// Blocchi bidimensionali contenenti tanti thread quanti sono i nodi in output
	dim3 threadBlocks(_height, _width, 1);

	// Tanti blocchi quanto il numero di filtri
	dim3 numBlocks(1, 1, _depth);

	Kernel::zeroPaddingBisK(numBlocks, threadBlocks, padding, error, _width, _filterWidth);

#ifdef DEBUG
	std::cout << "\n\nerror con zero padding\n\n";
	printFromCudaFormatted(padding, paddingSize, paddingWidth);
#endif

	// Dimensione insieme submatrici in byte = creo una submatrice per ogni nodo di output di L-1
	const int prevUniqueNodes = prevNodes / _prevLayerDepth;

	// Blocchi tridimensionali contenenti tanti thread quanti la grandezza dei filtri
	threadBlocks = dim3(_filterWidth, _filterWidth, 1);

	// Tanti blocchi quanti sono i nodi in input e il depth del livello precedente
	numBlocks = dim3(sqrt(prevUniqueNodes), sqrt(prevUniqueNodes), _prevLayerDepth);

	Kernel::createSubmatrixBisK(numBlocks, threadBlocks, subCalcError, padding, paddingWidth, _filterWidth, _stride, prevUniqueNodes);

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici zero padding\n\n";
	printFromCudaFormatted(sub, prevUniqueNodes * _depth * _filterDim, _filterWidth);
#endif

	// Ruoto subito i pesi aggiornati per poi usarli nella backpropagation al livello L-1
	// Blocchi bidimensionali contenenti tanti thread quanti il numero di filtri
	threadBlocks = dim3(_depth, _prevLayerDepth, 1);

	// Tanti blocchi quante sono le righe e le colonne di forwardError
	numBlocks = dim3(_filterWidth, _filterWidth, 1);

	Kernel::rot180BisK(numBlocks, threadBlocks, weight, weightRot, _filterDim);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore dei pesi ruotati\n\n";
	printFromCudaFormatted(weightRot, _wDim, _filterWidth);
#endif

	//ora sono in una situazione simile alla convoluzione
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {
			CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _filterDim, prevUniqueNodes, &alpha, subCalcError + (i * prevUniqueNodes), _filterDim, weightRot + ((i + j * _depth) * _filterDim), 1, &beta, prevError + (j * prevUniqueNodes), 1));
		}
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi back propagation\n\n";
	printFromCudaFormatted(prevError, prevNodes, sqrt(prevUniqueNodes));
#endif
}

void Batch::back_propagation_output(const double * prevOutput, const uint8_t * labels, const int & target, const double & learningRate) {
	// Calcolo dell'errore per ogni nodo
	Kernel::outputErrorK((_alignedNodes / THREADS), THREADS, output, error, labels, target, _nodes);

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nErrore commesso sui nodi back propagation output\n\n";
	printFromCudaFormatted(error, _nodes, _width);
#endif

	// Calcolo della Back Propagation
	calcBackPropagation(prevOutput, learningRate);
}

void Batch::back_propagation(const double *prevOutput, const double &learningRate) {
	calcBackPropagation(prevOutput, learningRate);
}

void Batch::calcBackPropagation(const double *prevOutput, const double &learningRate) {
	updateWeights(prevOutput, learningRate);
}

void Batch::updateWeights(const double *prevOutput, const double &learningRate) {

	// Blocchi tridimensionali contenenti tanti thread quanti sono i nodi in output
	dim3 threadBlocks(_width, _height, 1);

	// Tanti blocchi quanti la grandezza dei filtri e il depth del livello precedente
	dim3 numBlocks(_filterWidth, _filterWidth, _prevLayerDepth);

	Kernel::createSubmatrixBisK(numBlocks, threadBlocks, subBack, prevOutput, _prevLayerWidth, _width, _stride, _filterDim);

#ifdef DEBUG_SUB
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nValore submatrici backpropagation\n\n";
	printFromCudaFormatted(subBack, _uniqueNodes * _filterDim, _width);
#endif

	//ora sono in una situazione simile al fully connected
	double backAlpha = 1.0 / _uniqueNodes;
	for (int i = 0; i < _depth; i++) {
		for (int j = 0; j < _prevLayerDepth; j++) {
			CHECK_CUBLAS(hipblasDgemv(handle, HIPBLAS_OP_T, _uniqueNodes, _filterDim, &backAlpha, subBack + (j * _filterDim), _uniqueNodes, error + (i * _uniqueNodes), 1, &beta, tempWeight + ((i + j * _depth) * _filterDim), 1));
		}
	}

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice temporanea per aggiornamento pesi\n\n";
	printFromCudaFormatted(tempWeightRot, _wDim, _filterWidth);
#endif

	// Aggiornamento effettivo dei pesi 
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _wDim, &learningRate, tempWeight, 1, weight, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, _wDim, _depth, &learningRate, tempWeight, _wDim, &alpha, weight, _wDim, weight, _wDim));

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nMatrice dei pesi aggiornata\n\n";
	printFromCudaFormatted(weight, _wDim, _filterWidth);
#endif

	// Aggiornamento del bias 
	CHECK_CUBLAS(
		hipblasDaxpy(handle, _nodes, &learningRate, error, 1, bias, 1));
	//CHECK_CUBLAS(
		//hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, _nodes, &learningRate, errorRot, 1, &alpha, bias, 1, bias, 1));

#ifdef DEBUG
	CHECK(hipDeviceSynchronize());
	std::cout << "\n\nVettore del bias aggiornato\n\n";
	printFromCudaFormatted(bias, _nodes, _width);
#endif
}

void Batch::deleteCuda() {
	CHECK_CUBLAS(hipblasDestroy(handle));
	CHECK(hipFree(weight));
	CHECK(hipFree(weightRot));
	CHECK(hipFree(bias));
	CHECK(hipFree(output));
	CHECK(hipFree(error));
	CHECK(hipFree(tempWeight));
	CHECK(hipFree(tempOutput));
	CHECK(hipFree(subForward));
	CHECK(hipFree(subCalcError));
	CHECK(hipFree(subBack));
	CHECK(hipFree(padding));
}

void Batch::printW() {
	printFromCudaFormatted(weight, _wDim, _filterWidth);
	//printFromCudaFormatted(bias, _nodes, _width);
}

int Batch::_calcOutput(bool withPadding) {
	//PER ORA NON CONSIDERATO CASO IN CUI SI GENERANO ERRORI (padding numero non intero, filtro pi� grande dell'input, stride che non combacia, ecc)
	if (_filterWidth > _prevLayerWidth) {
		std::cerr << "Le dimensioni del filtro superano le dimensioni del livello precedente!!" << std::endl;
		exit(1);
	}

	if (withPadding) {
		_padding = (_filterWidth - 1) / 2;
		return _prevLayerWidth;
	}

	//+(_stride - 1)) serve per aggiornare per eccesso
	return ((_prevLayerWidth - _filterWidth + (_stride - 1)) / _stride) + 1;
}
