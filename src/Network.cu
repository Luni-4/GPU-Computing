// Cuda
#include <hip/hip_runtime.h>

#include <iostream>

#include "Common.h"
#include "Network.h"

Network::Network(const std::vector<std::unique_ptr<LayerDefinition>> &layers) {
    
    for(auto& l: layers)
		_layers.push_back(l.get());
}

Network::~Network() {    
    // Liberare la memoria del device
    CHECK(hipFree(inputImg));
    //CHECK(hipDeviceReset());
}

void Network::train(Data *data, const int &epoch, const double &eta, const double &lambda) {
    //Leggere i dati dal training set
    data->readTrainData();
    
    // Caricare i dati in Cuda
    cudaDataLoad(data);
    
    // Inizializzare le strutture della rete
    cudaInitStruct(data);
    
    // Numero di esempi nel training set
    const int nImages = data->getLabelSize();
    
    // Dimensione della singola immagine
    const int imgDim = data->getImgDimension();
    
    // Quantità di dati da allocare e copiare
    const int iBytes = imgDim * sizeof(double); 
	
	// Allocare il buffer di input della singola coppia (etichetta,immagine)
	CHECK(hipMalloc((void**)&inputImg, iBytes));
	
	int i = 0;
    
    // Elabora ogni immagine
    //for(int i = 0; i < nImages; i++)
    //{
        int imgIndex = i * imgDim;
        
        // Copia dell'immagine corrente nel buffer
        CHECK(hipMemcpy(inputImg, (cudaData + imgIndex), iBytes, hipMemcpyDeviceToDevice));        
         
        //for(int j = 0; j < epoch; j++)
        //{
            // Forward_propagation per ogni livello
            forwardPropagation();
            
            // Calcolo dell'errore per ogni livello
            //error();
            
            // Backward_propagation per ogni livello
           // backwardPropagation();
        //}   
    
    //}
    
    // cancellare i dati di train dal device
    CHECK(hipFree(cudaData));
    CHECK(hipFree(cudaLabels));
    
}


void Network::cudaDataLoad(Data *data) {
	const int dBytes = data->getDataSize() * sizeof(double);
	const int lBytes = data->getLabelSize() * sizeof(uint8_t);

	// Allocare le matrici
	CHECK(hipMalloc((void**)&cudaData, dBytes));
	CHECK(hipMalloc((void**)&cudaLabels, lBytes));
	
	// Passare i dati
	CHECK(hipMemcpy(cudaData, data->getCudaData(), dBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(cudaLabels, data->getCudaLabels(), lBytes, hipMemcpyHostToDevice));

	// Liberare i dati dalla CPU
	data->clearDataCPU();
}

void Network::cudaInitStruct(Data *data) {
    
    _layers[0]->defineCuda(data->getImgWidth(), data->getImgHeight(), data->getImgDepth());
    
    for(std::size_t i = 1; i < _layers.size(); i++)
    {
        const int prevWidth = _layers[i-1]->getWidth();
        const int prevHeight = _layers[i-1]->getHeight();
        const int prevDepth = _layers[i-1]->getDepth();
        
        _layers[i]->defineCuda(prevWidth, prevHeight, prevDepth);
    } 
}

void Network::forwardPropagation(){

    _layers[0]->forward_propagation(inputImg);   
    
    
    /*for(std::size_t i = 1; i < _layers.size(); i++)
    {       
        _layers[i]->forward_propagation(_layers[i-1].getOutput());
    }*/
}
